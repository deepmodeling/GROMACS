#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2019,2020,2021, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \internal \file
 *
 * \brief Implements PME-PP communication using CUDA
 *
 *
 * \author Alan Gray <alang@nvidia.com>
 *
 * \ingroup module_ewald
 */
#include "gmxpre.h"

#include "pme_pp_comm_gpu_impl.h"

#include "config.h"

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/device_context.h"
#include "gromacs/gpu_utils/device_stream.h"
#include "gromacs/gpu_utils/devicebuffer.h"
#include "gromacs/gpu_utils/gpueventsynchronizer.cuh"
#include "gromacs/gpu_utils/typecasts.cuh"
#include "gromacs/utility/gmxmpi.h"

namespace gmx
{

PmePpCommGpu::Impl::Impl(MPI_Comm             comm,
                         int                  pmeRank,
                         const DeviceContext& deviceContext,
                         const DeviceStream&  deviceStream) :
    deviceContext_(deviceContext),
    comm_(comm),
    pmeRank_(pmeRank),
    pmePpCommStream_(deviceStream)
{
}

PmePpCommGpu::Impl::~Impl() {}

void PmePpCommGpu::Impl::reinit(int size)
{
    // This rank will access PME rank memory directly, so needs to receive the remote PME buffer addresses.
#if GMX_MPI

    if (GMX_THREAD_MPI)
    {
        MPI_Recv(&remotePmeXBuffer_, sizeof(float3**), MPI_BYTE, pmeRank_, 0, comm_, MPI_STATUS_IGNORE);
        MPI_Recv(&remotePmeFBuffer_, sizeof(float3**), MPI_BYTE, pmeRank_, 0, comm_, MPI_STATUS_IGNORE);
    }

#endif

    // Reallocate buffer used for staging PME force on GPU
    reallocateDeviceBuffer(&d_pmeForces_, size, &d_pmeForcesSize_, &d_pmeForcesSizeAlloc_, deviceContext_);
    return;
}

void PmePpCommGpu::Impl::receiveForceFromPmeCudaDirect(float3* pmeForcePtr, int recvSize, bool receivePmeForceToGpu)
{
    // Receive event from PME task and add to stream, to ensure pull of data doesn't
    // occur before PME force calc is completed

#if GMX_MPI
    GpuEventSynchronizer* pmeSync;
    MPI_Recv(&pmeSync, sizeof(GpuEventSynchronizer*), MPI_BYTE, pmeRank_, 0, comm_, MPI_STATUS_IGNORE);
    pmeSync->enqueueWaitEvent(pmePpCommStream_);
#endif

    // Pull force data from remote GPU
    hipError_t stat = hipMemcpyAsync(pmeForcePtr,
                                       remotePmeFBuffer_,
                                       recvSize * DIM * sizeof(float),
                                       hipMemcpyDefault,
                                       pmePpCommStream_.stream());
    CU_RET_ERR(stat, "hipMemcpyAsync on Recv from PME CUDA direct data transfer failed");

    if (receivePmeForceToGpu)
    {
        // Record event to be enqueued in the GPU local buffer operations, to
        // satisfy dependency on receiving the PME force data before
        // reducing it with the other force contributions.
        forcesReadySynchronizer_.markEvent(pmePpCommStream_);
    }
    else
    {
        // Ensure CPU waits for PME forces to be copied before reducing
        // them with other forces on the CPU
        pmePpCommStream_.synchronize();
    }
}

void PmePpCommGpu::Impl::receiveForceFromPmeCudaMPI(float3* pmeForcePtr, int recvSize)
{
#if GMX_MPI
    // arbitrarily chosen
    const int tag = 101;

    MPI_Recv(pmeForcePtr, recvSize * DIM, MPI_FLOAT, pmeRank_, tag, comm_, MPI_STATUS_IGNORE);
#else
    GMX_UNUSED_VALUE(pmeForcePtr);
    GMX_UNUSED_VALUE(recvSize);
#endif
}

void PmePpCommGpu::Impl::receiveForceFromPme(float3* recvPtr, int recvSize, bool receivePmeForceToGpu)
{
    float3* pmeForcePtr = receivePmeForceToGpu ? d_pmeForces_ : recvPtr;
    if (GMX_THREAD_MPI)
    {
        receiveForceFromPmeCudaDirect(pmeForcePtr, recvSize, receivePmeForceToGpu);
    }
    else
    {
        receiveForceFromPmeCudaMPI(pmeForcePtr, recvSize);
    }
}

void PmePpCommGpu::Impl::sendCoordinatesToPmeCudaDirect(float3*               sendPtr,
                                                        int                   sendSize,
                                                        GpuEventSynchronizer* coordinatesReadyOnDeviceEvent)
{
    // ensure stream waits until coordinate data is available on device
    coordinatesReadyOnDeviceEvent->enqueueWaitEvent(pmePpCommStream_);

    hipError_t stat = hipMemcpyAsync(remotePmeXBuffer_,
                                       sendPtr,
                                       sendSize * DIM * sizeof(float),
                                       hipMemcpyDefault,
                                       pmePpCommStream_.stream());
    CU_RET_ERR(stat, "hipMemcpyAsync on Send to PME CUDA direct data transfer failed");

#if GMX_MPI
    // Record and send event to allow PME task to sync to above transfer before commencing force calculations
    pmeCoordinatesSynchronizer_.markEvent(pmePpCommStream_);

    GpuEventSynchronizer* pmeSync = &pmeCoordinatesSynchronizer_;
    MPI_Send(&pmeSync, sizeof(GpuEventSynchronizer*), MPI_BYTE, pmeRank_, 0, comm_);
#endif
}

void PmePpCommGpu::Impl::sendCoordinatesToPmeCudaMPI(float3*               sendPtr,
                                                     int                   sendSize,
                                                     GpuEventSynchronizer* coordinatesReadyOnDeviceEvent)
{
    // ensure coordinate data is available on device before we start transfer
    coordinatesReadyOnDeviceEvent->waitForEvent();

#if GMX_MPI
    float3* sendptr_x = sendPtr;

    MPI_Send(sendptr_x, sendSize * DIM, MPI_FLOAT, pmeRank_, 0, comm_);
#else
    GMX_UNUSED_VALUE(sendPtr);
    GMX_UNUSED_VALUE(sendSize);
#endif
}

void PmePpCommGpu::Impl::sendCoordinatesToPme(float3*               sendPtr,
                                              int                   sendSize,
                                              GpuEventSynchronizer* coordinatesReadyOnDeviceEvent)
{
    if (GMX_THREAD_MPI)
    {
        sendCoordinatesToPmeCudaDirect(sendPtr, sendSize, coordinatesReadyOnDeviceEvent);
    }
    else
    {
        sendCoordinatesToPmeCudaMPI(sendPtr, sendSize, coordinatesReadyOnDeviceEvent);
    }
}
void* PmePpCommGpu::Impl::getGpuForceStagingPtr()
{
    return static_cast<void*>(d_pmeForces_);
}

GpuEventSynchronizer* PmePpCommGpu::Impl::getForcesReadySynchronizer()
{
    if (GMX_THREAD_MPI)
    {
        return &forcesReadySynchronizer_;
    }
    else
    {
        return nullptr;
    }
}

PmePpCommGpu::PmePpCommGpu(MPI_Comm             comm,
                           int                  pmeRank,
                           const DeviceContext& deviceContext,
                           const DeviceStream&  deviceStream) :
    impl_(new Impl(comm, pmeRank, deviceContext, deviceStream))
{
}

PmePpCommGpu::~PmePpCommGpu() = default;

void PmePpCommGpu::reinit(int size)
{
    impl_->reinit(size);
}

void PmePpCommGpu::receiveForceFromPme(RVec* recvPtr, int recvSize, bool receivePmeForceToGpu)
{
    impl_->receiveForceFromPme(asFloat3(recvPtr), recvSize, receivePmeForceToGpu);
}

void PmePpCommGpu::sendCoordinatesToPmeFromGPU(DeviceBuffer<RVec>    sendPtr,
                                               int                   sendSize,
                                               GpuEventSynchronizer* coordinatesReadyOnDeviceEvent)
{
    impl_->sendCoordinatesToPme(asFloat3(sendPtr), sendSize, coordinatesReadyOnDeviceEvent);
}

void PmePpCommGpu::sendCoordinatesToPmeFromCPU(RVec*                 sendPtr,
                                               int                   sendSize,
                                               GpuEventSynchronizer* coordinatesReadyOnDeviceEvent)
{
    impl_->sendCoordinatesToPme(asFloat3(sendPtr), sendSize, coordinatesReadyOnDeviceEvent);
}

void* PmePpCommGpu::getGpuForceStagingPtr()
{
    return impl_->getGpuForceStagingPtr();
}

GpuEventSynchronizer* PmePpCommGpu::getForcesReadySynchronizer()
{
    return impl_->getForcesReadySynchronizer();
}

} // namespace gmx
