#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2012,2013,2014,2015,2016,2017,2018,2019, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \file
 *  \brief Define CUDA implementation of sits_gpu.h
 *
 *  \author Junhan Chang <changjh@pku.edu.cn>
 */
#include "gmxpre.h"

#include "config.h"

#include <assert.h>
#include <stdlib.h>

#include "gromacs/sits/sits_gpu.h"

#if defined(_MSVC)
#    include <limits>
#endif

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/gpueventsynchronizer.cuh"
#include "gromacs/gpu_utils/vectype_ops.cuh"
#include "gromacs/mdtypes/simulation_workload.h"

#include "gromacs/sits/sits_gpu_data_mgmt.h"
#include "gromacs/sits/sits.h"

#include "gromacs/timing/gpu_timing.h"
#include "gromacs/utility/cstringutil.h"
#include "gromacs/utility/gmxassert.h"

#include "gromacs/math/utilities.h"

#include "sits_cuda_types.h"

#define FLT_MAX 10e8;

/*! As we execute nonbonded workload in separate streams, before launching
   the kernel we need to make sure that he following operations have completed:
   - atomdata allocation and related H2D transfers (every nstlist step);
   - pair list H2D transfer (every nstlist step);
   - shift vector H2D transfer (every nstlist step);
   - force (+shift force and energy) output clearing (every step).

   These operations are issued in the local stream at the beginning of the step
   and therefore always complete before the local kernel launch. The non-local
   kernel is launched after the local on the same device/context hence it is
   inherently scheduled after the operations in the local stream (including the
   above "misc_ops") on pre-GK110 devices with single hardware queue, but on later
   devices with multiple hardware queues the dependency needs to be enforced.
   We use the misc_ops_and_local_H2D_done event to record the point where
   the local x+q H2D (and all preceding) tasks are complete and synchronize
   with this event in the non-local stream before launching the non-bonded kernel.
 */
// void gpu_enhance_force(gmx_sits_cuda_t* gpu_sits)
// {
//     cu_sits_atdat_t* atdat  = gpu_sits->sits_atdat;
//     cu_sits_param_t* param  = gpu_sits->sits_param;
//     hipStream_t     stream = *(gpu_sits->stream);

//     /* Kernel launch config:
//      * - The thread block dimensions match the size of i-clusters, j-clusters,
//      *   and j-cluster concurrency, in x, y, and z, respectively.
//      * - The 1D block-grid contains as many blocks as super-clusters.
//      */
//     int num_threads_z = 1;
//     if (nb->dev_info->prop.major == 3 && nb->dev_info->prop.minor == 7)
//     {
//         num_threads_z = 2;
//     }
//     int nblock = calc_nb_kernel_nblock(plist->nsci, nb->dev_info);


//     KernelLaunchConfig config;
//     config.blockSize[0]     = c_clSize;
//     config.blockSize[1]     = c_clSize;
//     config.blockSize[2]     = num_threads_z;
//     config.gridSize[0]      = nblock;
//     config.sharedMemorySize = calc_shmem_required_nonbonded(num_threads_z, nb->dev_info, nbp);
//     config.stream           = stream;

//     if (debug)
//     {
//         fprintf(debug,
//                 "Non-bonded GPU launch configuration:\n\tThread block: %zux%zux%zu\n\t"
//                 "\tGrid: %zux%zu\n\t#Super-clusters/clusters: %d/%d (%d)\n"
//                 "\tShMem: %zu\n",
//                 config.blockSize[0], config.blockSize[1], config.blockSize[2], config.gridSize[0],
//                 config.gridSize[1], plist->nsci * c_numClPerSupercl, c_numClPerSupercl, plist->na_c,
//                 config.sharedMemorySize);
//     }

//     auto*      timingEvent = bDoTime ? t->interaction[iloc].nb_k.fetchNextEvent() : nullptr;
//     const auto kernel      = select_nbnxn_kernel(
//             nbp->eeltype, nbp->vdwtype, stepWork.computeEnergy,
//             (plist->haveFreshList && !nb->timers->interaction[iloc].didPrune), nb->dev_info);
//     const auto kernelArgs =
//             prepareGpuKernelArguments(kernel, config, adat, nbp, plist, &stepWork.computeVirial);
//     launchGpuKernel(kernel, config, timingEvent, "k_calc_nb", kernelArgs);

//     if (GMX_NATIVE_WINDOWS)
//     {
//         /* Windows: force flushing WDDM queue */
//         hipStreamQuery(stream);
//     }
// }

static __device__ __host__ float log_add_log(float a, float b)
{
    return fmaxf(a, b) + logf(1.0 + expf(-fabsf(a - b)));
}

static __global__ void Sits_Record_Ene(float*       ene_record,
                                       const float* pw_ene,
                                       const float* pp_ene,
                                       const float  pe_a,
                                       const float  pe_b,
                                       const float  pw_factor)
{
    float temp = *pw_ene * pw_factor + *pp_ene;
    temp       = pe_a * temp + pe_b;

    *ene_record = temp;
    // printf("DEBUG ene_record: %f\n", ene_record[0]);
}

static __global__ void Sits_Update_gf(const int kn, float* gf, const float* ene_record, const float* log_nk, const float* beta_k)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < kn)
    {
        gf[i] = -beta_k[i] * ene_record[0] + log_nk[i];
        // printf("DEBUG gf: %d %f\n", i, gf[i]);
    }
}

static __global__ void Sits_Update_gfsum(const int kn, float* gfsum, const float* gf)
{
    if (threadIdx.x == 0)
    {
        gfsum[0] = -FLT_MAX;
    }
    for (int i = 0; i < kn; i = i + 1)
    {
        gfsum[0] = log_add_log(gfsum[0], gf[i]);
        // printf("DEBUG gfsum: %d %f %f\n", i, gfsum[0], gf[i]);
    }
}

static __global__ void Sits_Update_log_pk(const int kn, float* log_pk, const float* gf, const float* gfsum, const int reset)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < kn)
    {
        if (reset == 1)
        {
            log_pk[i] = -FLT_MAX;
        }
        float gfi = gf[i];
        log_pk[i] = log_add_log(log_pk[i], gfi - gfsum[0]);
        // printf("DEBUG log_pk: %d %f %f\n", i, log_pk[i], gfsum[0]);
    }
}

static __global__ void Sits_Update_log_mk_inv(const int    kn,
                                              float*       log_weight,
                                              float*       log_mk_inv,
                                              float*       log_norm_old,
                                              float*       log_norm,
                                              const float* log_pk,
                                              const float* log_nk)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < kn - 1)
    {
        log_weight[i] = (log_pk[i] + log_pk[i + 1]) * 0.5;
        // printf("DEBUG log_weight: %d %f %f\n", i, log_pk[i], log_pk[i + 1]);
        log_mk_inv[i]   = log_nk[i] - log_nk[i + 1];
        log_norm_old[i] = log_norm[i];
        log_norm[i]     = log_add_log(log_norm[i], log_weight[i]);
        log_mk_inv[i] =
                log_add_log(log_mk_inv[i] + log_norm_old[i] - log_norm[i],
                            log_pk[i + 1] - log_pk[i] + log_mk_inv[i] + log_weight[i] - log_norm[i]);
        // printf("DEBUG log_norm: %d %f %f\n", i, log_norm[i], log_weight[i]);
    }
}

static __global__ void Sits_Update_log_nk_inv(const int kn, float* log_nk_inv, const float* log_mk_inv)
{
    for (int i = 0; i < kn - 1; i++)
    {
        log_nk_inv[i + 1] = log_nk_inv[i] + log_mk_inv[i];
    }
}

static __global__ void Sits_Update_nk(const int kn, float* log_nk, float* nk, const float* log_nk_inv)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < kn)
    {
        log_nk[i] = -log_nk_inv[i];
        nk[i]     = exp(log_nk[i]);
    }
}

__global__ void sits_enhance_force_Calculate_nkExpBetakU_1(const int    k_numbers,
                                                                const float* beta_k,
                                                                const float* nk,
                                                                float*       nkexpbetaku,
                                                                const float  ene)
{
    float lin = beta_k[k_numbers - 1];
    for (int i = threadIdx.x; i < k_numbers; i = i + blockDim.x)
    {
        nkexpbetaku[i] = nk[i] * expf(-(beta_k[i] - lin) * ene);
        // printf("%f %f\n", beta_k[i], nkexpbetaku[i]);
    }
}

__global__ void sits_enhance_force_Calculate_nkExpBetakU_2(const int    k_numbers,
                                                                const float* beta_k,
                                                                const float* nk,
                                                                float*       nkexpbetaku,
                                                                const float  ene)
{
    float lin = beta_k[0];
    for (int i = threadIdx.x; i < k_numbers; i = i + blockDim.x)
    {
        nkexpbetaku[i] = nk[i] * expf(-(beta_k[i] - lin) * ene);
        // printf("%f %f\n", beta_k[i], nkexpbetaku[i]);
    }
}

__global__ void sits_enhance_force_Sum_Of_Above(const int    k_numbers,
                                                     const float* nkexpbetaku,
                                                     const float* beta_k,
                                                     float*       sum_of_above)
{
    if (threadIdx.x == 0)
    {
        sum_of_above[0] = 0.;
    }
    __syncthreads();
    float lin = 0.;
    for (int i = threadIdx.x; i < k_numbers; i = i + blockDim.x)
    {
        lin = lin + beta_k[i] * nkexpbetaku[i];
    }
    atomicAdd(sum_of_above, lin);
}

__global__ void sits_enhance_force_Sum_Of_nkExpBetakU(const int    k_numbers,
                                                           const float* nkexpbetaku,
                                                           float*       sum_of_below)
{
    if (threadIdx.x == 0)
    {
        sum_of_below[0] = 0.;
    }
    __syncthreads();
    float lin = 0.;
    for (int i = threadIdx.x; i < k_numbers; i = i + blockDim.x)
    {
        lin = lin + nkexpbetaku[i];
        // printf("%f\n", nkexpbetaku[i]);
    }
    atomicAdd(sum_of_below, lin);
}

__global__ void sits_enhance_force_update_factor(float*        sum_a,
                                                float*        sum_b,
                                                float*        factor,
                                                const float   beta_0,
                                                const float   fb_bias)
{
    if (threadIdx.x == 0)
    {
        if (isinf(factor[0]) || isnan(factor[0]) || factor[0] == 0.0)
        {
            factor[0] = 1.0;
        }
        if (isinf(factor[1]) || isnan(factor[1]) || factor[1] == 0.0)
        {
            factor[1] = 1.0;
        }
        factor[0] = sum_a[0] / sum_b[0] / beta_0 + fb_bias;
        // avoid crashing caused by sharp fluctuation of fc_ball
        if (!isinf(factor[0]) && !isnan(factor[0]) && (factor[0] > 0.4 * factor[1])
            && (factor[0] < 2 * factor[1]))
        {
            factor[1] = factor[0];
        }
        else
        {
            factor[0] = factor[1];
        }
    }
    // printf("\n| sum_a | sum_b | factor | factor1 |\n");
    // printf(" %7.3f %7.3f %8.3f %8.3f \n", *sum_a, *sum_b, factor[0], factor[1]);
}

static __global__ void sits_enhance_force_Protein(const int     protein_numbers,
                                                  float3*       md_frc,
                                                  const float3* pw_frc,
                                                  const float   fc_ball,
                                                  const float   pw_factor)
{
    for (int i = threadIdx.x; i < protein_numbers; i = i + blockDim.x)
    {
        md_frc[i].x = fc_ball * (md_frc[i].x) + pw_factor * pw_frc[i].x;
        md_frc[i].y = fc_ball * (md_frc[i].y) + pw_factor * pw_frc[i].y;
        md_frc[i].z = fc_ball * (md_frc[i].z) + pw_factor * pw_frc[i].z;
    }
}

static __global__ void sits_enhance_force_Water(const int     protein_numbers,
                                                const int     natoms,
                                                float3*       md_frc,
                                                const float3* pw_frc,
                                                const float   pw_factor)
{
    for (int i = threadIdx.x + protein_numbers; i < natoms; i = i + blockDim.x)
    {
        md_frc[i].x = md_frc[i].x + pw_factor * pw_frc[i].x;
        md_frc[i].y = md_frc[i].y + pw_factor * pw_frc[i].y;
        md_frc[i].z = md_frc[i].z + pw_factor * pw_frc[i].z;
    }
}

static __global__ void sits_enhance_force_by_energrp(const int     natoms,
                                                     int*          energrp,
                                                     float3*       md_frc,
                                                     const float3* pw_frc,
                                                     float*        fc_ball,
                                                     const float   pw_factor)
{
    for (int i = threadIdx.x; i < natoms; i = i + blockDim.x)
    {
        float fc_1 = fc_ball[0] - 1.0;
        if (energrp[i] == 0)
        {
            md_frc[i] *= fc_1;
        }
        else
        {
            md_frc[i] = make_float3(0.0);
        }
        md_frc[i] += fc_1 * pw_factor * pw_frc[i];
    }
}

/*-------------------------------- End CUDA kernels-----------------------------*/

void Sits_Classical_Enhance_Force(const int     natoms,
                                              int*          energrp,
                                              const float   pw_factor,
                                              float3*       md_frc,
                                              const float3* pw_frc,
                                              const float*  pp_ene,
                                              const float*  pw_ene,
                                              const int     k_numbers,
                                              float*        nkexpbetaku,
                                              const float*  beta_k,
                                              const float*  n_k,
                                              float*        sum_a,
                                              float*        sum_b,
                                              float*        factor,
                                              const float   beta_0,
                                              const float   pe_a,
                                              const float   pe_b,
                                              const float   fb_bias)
{
    float* h_E_pp;
    float* h_E_pw;

    h_E_pp = (float *) malloc(sizeof(float));
    h_E_pw = (float *) malloc(sizeof(float));
    hipMemcpy(h_E_pp, pp_ene, sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_E_pw, pw_ene, sizeof(float), hipMemcpyDeviceToHost);
    float ene = *(h_E_pp) + pw_factor * *(h_E_pw);
    ene       = pe_a * ene + pe_b;
    if (ene > 0)
    {
        sits_enhance_force_Calculate_nkExpBetakU_1<<<1, 64>>>(k_numbers, beta_k, n_k,
                                                                   nkexpbetaku, ene);
    }
    else
    {
        sits_enhance_force_Calculate_nkExpBetakU_2<<<1, 64>>>(k_numbers, beta_k, n_k,
                                                                   nkexpbetaku, ene);
    }

    sits_enhance_force_Sum_Of_nkExpBetakU<<<1, 128>>>(k_numbers, nkexpbetaku, sum_b);

    sits_enhance_force_Sum_Of_Above<<<1, 128>>>(k_numbers, nkexpbetaku, beta_k, sum_a);

    sits_enhance_force_update_factor<<<1, 1>>>(sum_a, sum_b, factor, beta_0, fb_bias);
    //	printf("factor %e sum0 %e %e ene %f lfactor %e\n", fc, sum_a[0], sum_b[0], ene, factor[1]);

    // line
    // fc = (ene - 20.) / 80./2. + 0.2;
    sits_enhance_force_by_energrp<<<32, 128>>>(natoms, energrp, md_frc, pw_frc, factor, pw_factor);
}

namespace Sits
{

void gpu_update_params(gmx_sits_cuda_t* gpu_sits, int step, FILE* nklog, FILE* normlog)
{
    cu_sits_atdat_t* atdat = gpu_sits->sits_atdat;
    cu_sits_param_t* param = gpu_sits->sits_param;

    if (!param->constant_nk && step % param->record_interval == 0)
    {
        Sits_Record_Ene<<<1, 1>>>(param->ene_recorded, &(atdat->d_enerd[1]), &(atdat->d_enerd[0]),
                                  param->energy_multiple, param->energy_shift, atdat->pw_enh_factor);

        Sits_Update_gf<<<ceilf((float)param->k_numbers / 32.), 32>>>(
                param->k_numbers, param->gf, param->ene_recorded,
                param->log_nk, param->beta_k);

        Sits_Update_gfsum<<<1, 1>>>(param->k_numbers, param->gfsum, param->gf);

        Sits_Update_log_pk<<<ceilf((float)param->k_numbers / 32.), 32>>>(
                param->k_numbers, param->log_pk, param->gf,
                param->gfsum, param->reset);

        param->reset = 0;
        param->record_count++;

        if ((param->record_count % param->update_interval == 0) && (param->record_count / param->update_interval < param->niter))
        {
            Sits_Update_log_mk_inv<<<ceilf((float)param->k_numbers / 32.), 32>>>(
                    param->k_numbers, param->log_weight, param->log_mk_inv,
                    param->log_norm_old, param->log_norm, param->log_pk,
                    param->log_nk);

            Sits_Update_log_nk_inv<<<1, 1>>>(param->k_numbers, param->log_nk_inv,
                                             param->log_mk_inv);

            Sits_Update_nk<<<ceilf((float)param->k_numbers / 32.), 32>>>(
                    param->k_numbers, param->log_nk, param->nk,
                    param->log_nk_inv);

            // param->record_count = 0;
            param->reset        = 1;

            if (!param->constant_nk)
            {
                float* h_log_nk;
                h_log_nk = (float*) malloc(param->k_numbers * sizeof(float));
                hipMemcpy(h_log_nk, param->log_nk, sizeof(float) * param->k_numbers, hipMemcpyDeviceToHost);

                float* h_log_pk;
                h_log_pk = (float*) malloc(param->k_numbers * sizeof(float));
                hipMemcpy(h_log_pk, param->log_pk, sizeof(float) * param->k_numbers, hipMemcpyDeviceToHost);

                float* h_log_norm;
                h_log_norm = (float*) malloc(param->k_numbers * sizeof(float));
                hipMemcpy(h_log_norm, param->log_norm, sizeof(float) * param->k_numbers, hipMemcpyDeviceToHost);

                if (nklog)
                {
                    for (int i = 0; i < param->k_numbers; i++){
                        fprintf(nklog, "%8.4f ", h_log_nk[i]);
                    }
                    for (int i = 0; i < param->k_numbers; i++){
                        fprintf(nklog, "%8.4f ", h_log_pk[i]);
                    }
                    fprintf(nklog, "\n");
                }

                if (normlog)
                {
                    for (int i = 0; i < param->k_numbers; i++){
                        fprintf(normlog, "%8.4f ", h_log_norm[i]);
                    }
                    fprintf(nklog, "\n");
                }
                // hipMemcpy(param->log_nk_recorded_cpu, param->nk,
                //            sizeof(float) * param->k_numbers, hipMemcpyDeviceToHost);
                // fwrite(param->log_nk_recorded_cpu, sizeof(float), param->k_numbers,
                //        param->nk_traj_file);
                // hipMemcpy(param->log_norm_recorded_cpu, param->log_norm,
                //            sizeof(float) * param->k_numbers, hipMemcpyDeviceToHost);
                // fwrite(param->log_norm_recorded_cpu, sizeof(float),
                //        param->k_numbers, param->norm_traj_file);
            }
        }
    }
}

void gpu_enhance_force(gmx_sits_cuda_t* gpu_sits, int step)
{
    cu_sits_atdat_t* atdat = gpu_sits->sits_atdat;
    cu_sits_param_t* param = gpu_sits->sits_param;

    if (atdat->sits_cal_mode == 0)
    {
        Sits_Classical_Enhance_Force(
                atdat->natoms, atdat->energrp, atdat->pw_enh_factor, 
                atdat->d_force_tot_nbat, atdat->d_force_pw_nbat, 
                &(atdat->d_enerd[0]), &(atdat->d_enerd[1]),
                param->k_numbers, param->nkExpBetakU, param->beta_k,
                param->nk, param->sum_a, param->sum_b,
                param->factor, param->beta0, param->energy_multiple,
                param->energy_shift, param->fb_shift);
    }
    else if (atdat->sits_cal_mode == 1)
    {
        // Get fc_ball by random walk in given potential to reach certain marginal distribution
        // if (!simple_param->is_constant_fc_ball)
        // {
        //     fc_ball_random_walk();
        // }
        // else
        // {
        //     param->fc_ball = simple_param->constant_fc_ball;
        // }
        // sits_enhance_force_Protein<<<1, 128>>>(
        //         param->protein_natoms, frc, protein_water_frc, param->fc_ball,
        //         param->pwwp_enhance_factor * param->fc_ball + 1.0 - param->pwwp_enhance_factor);
        // sits_enhance_force_Water<<<1, 128>>>(
        //         param->protein_natoms, param->natoms, frc, protein_water_frc,
        //         param->pwwp_enhance_factor * param->fc_ball + 1.0 - param->pwwp_enhance_factor);
    }
    else
    {
        // sits_enhance_force_Protein<<<1, 128>>>(
        //         param->protein_natoms, frc, protein_water_frc, param->fc_ball,
        //         param->pwwp_enhance_factor * param->fc_ball + 1.0 - param->pwwp_enhance_factor);
        // sits_enhance_force_Water<<<1, 128>>>(
        //         param->protein_natoms, param->natoms, frc, protein_water_frc,
        //         param->pwwp_enhance_factor * param->fc_ball + 1.0 - param->pwwp_enhance_factor);
    }
}

// void sits_t::CLASSICAL_Sits_INFORMATION::Export_Restart_Information_To_File()
// {
//     FILE* nk;
//     Open_File_Safely(&nk, nk_rest_file, "w");
//     hipMemcpy(log_nk_recorded_cpu, nk, sizeof(float) * k_numbers, hipMemcpyDeviceToHost);
//     for (int i = 0; i < k_numbers; i++)
//     {
//         fprintf(nk, "%f\n", log_nk_recorded_cpu[i]);
//     }
//     fclose(nk);

//     FILE* norm;
//     Open_File_Safely(&norm, norm_rest_file, "w");
//     hipMemcpy(log_norm_recorded_cpu, log_norm, sizeof(float) * k_numbers, hipMemcpyDeviceToHost);
//     for (int i = 0; i < k_numbers; i++)
//     {
//         fprintf(norm, "%f\n", log_norm_recorded_cpu[i]);
//     }
//     fclose(norm);
// }

} // namespace Sits