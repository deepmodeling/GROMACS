#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2012,2013,2014,2015,2016,2017,2018,2019, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \file
 *  \brief Define CUDA implementation of nbnxn_gpu_data_mgmt.h
 *
 *  \author Szilard Pall <pall.szilard@gmail.com>
 */
#include "gmxpre.h"

#include <assert.h>
#include <stdarg.h>
#include <stdio.h>
#include <stdlib.h>

// TODO We would like to move this down, but the way gmx_nbnxn_gpu_t
//      is currently declared means this has to be before gpu_types.h
#include "gromacs/nbnxm/cuda/nbnxm_cuda_types.h"

// TODO Remove this comment when the above order issue is resolved
#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/gpu_utils.h"
#include "gromacs/gpu_utils/gpueventsynchronizer.cuh"
#include "gromacs/gpu_utils/pmalloc_cuda.h"
#include "gromacs/hardware/gpu_hw_info.h"
#include "gromacs/math/vectypes.h"
#include "gromacs/mdlib/force_flags.h"
#include "gromacs/mdtypes/interaction_const.h"
#include "gromacs/mdtypes/md_enums.h"
#include "gromacs/nbnxm/atomdata.h"
#include "gromacs/nbnxm/gpu_data_mgmt.h"
#include "gromacs/nbnxm/gridset.h"
#include "gromacs/nbnxm/nbnxm.h"
#include "gromacs/nbnxm/nbnxm_gpu.h"
#include "gromacs/nbnxm/pairlistsets.h"
#include "gromacs/pbcutil/ishift.h"
#include "gromacs/timing/gpu_timing.h"
#include "gromacs/utility/basedefinitions.h"
#include "gromacs/utility/cstringutil.h"
#include "gromacs/utility/fatalerror.h"
#include "gromacs/utility/real.h"
#include "gromacs/utility/smalloc.h"

#include "gromacs/nbnxm/cuda/nbnxm_cuda.h"

#include "sits_cuda_types.h"
#include "gromacs/sits/sits.h"
#include "gromacs/sits/sits_gpu_data_mgmt.h"

struct sits_atomdata_t;

namespace Sits
{

/* Fw. decl. */
static void sits_cuda_clear_e_fshift(gmx_sits_cuda_t* gpu_sits);

/*! Initializes the atomdata structure first time, it only gets filled at
    pair-search. */
static void sits_init_atomdata_first(cu_sits_atdat_t* atdat)
{
    hipError_t stat;

    stat = hipMalloc((void**)&atdat->d_enerd, 3 * sizeof(*atdat->d_enerd));
    CU_RET_ERR(stat, "hipMalloc failed on atdat->d_enerd");

    /* initialize to nullptr pointers to data that is not allocated here and will
       need reallocation in sits_cuda_init_atomdata */
    atdat->d_force_tot = nullptr;
    atdat->d_force_pw  = nullptr;
    atdat->d_force_tot_nbat = nullptr;
    atdat->d_force_pw_nbat  = nullptr;

    /* size -1 indicates that the respective array hasn't been initialized yet */
    atdat->natoms = -1;
    atdat->nalloc = -1;
}

/*! Initializes simulation constant data. */
static void cuda_init_sits_params(gmx_sits_cuda_t*           gpu_sits,
                                  const sits_atomdata_t*     sits_at)
{
    hipError_t    stat;
    cu_sits_param_t* param = gpu_sits->sits_param;
    hipStream_t stream    = *(gpu_sits->stream);

    sits_init_atomdata_first(gpu_sits->sits_atdat);

    // SITS ensemble definition
    param->record_interval = sits_at->record_interval;   // interval of energy record
    param->update_interval = sits_at->update_interval; // interval of $n_k$ update
    param->niter           = sits_at->niter;
    param->constant_nk     = sits_at->constant_nk;   // whether iteratively update n_k
    param->k_numbers       = sits_at->k_numbers;
    param->beta0           = sits_at->beta0;

    //计算时，可以对fc_ball直接修正，+ fb_shift进行调节，
    param->fb_shift        = sits_at->fb_shift;
    // energy record modifications: energy_record = energy_multiple * U + energy_shift;
    param->energy_multiple = sits_at->energy_multiple;
    param->energy_shift    = sits_at->energy_shift;

    // Derivations and physical quantities see:
    // \ref A selective integrated tempering method
    // \ref Self-adaptive enhanced sampling in the energy and trajectory spaces : Accelerated thermodynamics and kinetic calculations

    DeviceContext context = nullptr;

    param->k_nalloc = 0;
    reallocateDeviceBuffer(&param->beta_k, sits_at->k_numbers, &param->k_numbers, &param->k_nalloc, context);
    copyToDeviceBuffer(&param->beta_k, sits_at->beta_k.data(), 0, sits_at->k_numbers, stream,
                       GpuApiCallBehavior::Async, nullptr);
    
    param->k_nalloc = 0;
    reallocateDeviceBuffer(&param->nkExpBetakU, sits_at->k_numbers, &param->k_numbers, &param->k_nalloc, context);
    copyToDeviceBuffer(&param->nkExpBetakU, sits_at->nkExpBetakU.data(), 0, sits_at->k_numbers, stream,
                       GpuApiCallBehavior::Async, nullptr);
    
    param->k_nalloc = 0;
    reallocateDeviceBuffer(&param->nk, sits_at->k_numbers, &param->k_numbers, &param->k_nalloc, context);
    copyToDeviceBuffer(&param->nk, sits_at->nk.data(), 0, sits_at->k_numbers, stream,
                       GpuApiCallBehavior::Async, nullptr);
    
    stat = hipMalloc((void**)&param->sum_a, sizeof(*param->sum_a));
    CU_RET_ERR(stat, "hipMalloc failed on param->sum_a");
    stat = hipMalloc((void**)&param->sum_b, sizeof(*param->sum_b));
    CU_RET_ERR(stat, "hipMalloc failed on param->sum_b");
    stat = hipMalloc((void**)&param->factor, 2 * sizeof(*param->factor));
    CU_RET_ERR(stat, "hipMalloc failed on param->factor");

    stat = hipMemsetAsync(param->sum_a, 0, sizeof(*param->sum_a), stream);
    CU_RET_ERR(stat, "hipMemsetAsync on param->sum_a failed");
    stat = hipMemsetAsync(param->sum_b, 0, sizeof(*param->sum_b), stream);
    CU_RET_ERR(stat, "hipMemsetAsync on param->sum_b failed");
    stat = hipMemsetAsync(param->factor, 0, 2 * sizeof(*param->factor), stream);
    CU_RET_ERR(stat, "hipMemsetAsync on param->factor failed");

    // Details of $n_k$ iteration see:
    // \ref An integrate-over-temperature approach for enhanced sampling

    // |   .cpp var    |  ylj .F90 var  |  Ref var
    // | ene_recorded  | vshift         | U  
    // | gf            | gf             | log( n_k * exp(-beta_k * U) )
    // | gfsum         | gfsum          | log( Sum_(k=1)^N ( log( n_k * exp(-beta_k * U) ) ) )
    // | log_weight    | rb             | log of the weighting function
    // | log_mk_inv    | ratio          | log(m_k^-1)
    // | log_norm_old  | normlold       | W(j-1)
    // | log_norm      | norml          | W(j)
    // | log_pk        | rbfb           | log(p_k)
    // | log_nk_inv    | pratio         | log(n_k^-1)
    // | log_nk        | fb             | log(n_k)

    stat = hipMalloc((void**)&param->ene_recorded, sizeof(*param->ene_recorded));
    CU_RET_ERR(stat, "hipMalloc failed on param->ene_recorded");
    stat = hipMalloc((void**)&param->gfsum, sizeof(*param->gfsum));
    CU_RET_ERR(stat, "hipMalloc failed on param->gfsum");

    param->k_nalloc = 0;
    reallocateDeviceBuffer(&param->gf, sits_at->k_numbers, &param->k_numbers, &param->k_nalloc, context);
    copyToDeviceBuffer(&param->gf, sits_at->gf.data(), 0, sits_at->k_numbers, stream,
                       GpuApiCallBehavior::Async, nullptr);
    
    param->k_nalloc = 0;
    reallocateDeviceBuffer(&param->log_weight, sits_at->k_numbers, &param->k_numbers, &param->k_nalloc, context);
    copyToDeviceBuffer(&param->log_weight, sits_at->log_weight.data(), 0, sits_at->k_numbers, stream,
                       GpuApiCallBehavior::Async, nullptr);
    
    param->k_nalloc = 0;
    reallocateDeviceBuffer(&param->log_mk_inv, sits_at->k_numbers, &param->k_numbers, &param->k_nalloc, context);
    copyToDeviceBuffer(&param->log_mk_inv, sits_at->log_mk_inv.data(), 0, sits_at->k_numbers, stream,
                       GpuApiCallBehavior::Async, nullptr);
    
    param->k_nalloc = 0;
    reallocateDeviceBuffer(&param->log_norm_old, sits_at->k_numbers, &param->k_numbers, &param->k_nalloc, context);
    copyToDeviceBuffer(&param->log_norm_old, sits_at->log_norm_old.data(), 0, sits_at->k_numbers, stream,
                       GpuApiCallBehavior::Async, nullptr);
    
    param->k_nalloc = 0;
    reallocateDeviceBuffer(&param->log_norm, sits_at->k_numbers, &param->k_numbers, &param->k_nalloc, context);
    copyToDeviceBuffer(&param->log_norm, sits_at->log_norm.data(), 0, sits_at->k_numbers, stream,
                       GpuApiCallBehavior::Async, nullptr);
    
    param->k_nalloc = 0;
    reallocateDeviceBuffer(&param->log_pk, sits_at->k_numbers, &param->k_numbers, &param->k_nalloc, context);
    copyToDeviceBuffer(&param->log_pk, sits_at->log_pk.data(), 0, sits_at->k_numbers, stream,
                       GpuApiCallBehavior::Async, nullptr);
    
    param->k_nalloc = 0;
    reallocateDeviceBuffer(&param->log_nk_inv, sits_at->k_numbers, &param->k_numbers, &param->k_nalloc, context);
    copyToDeviceBuffer(&param->log_nk_inv, sits_at->log_nk_inv.data(), 0, sits_at->k_numbers, stream,
                       GpuApiCallBehavior::Async, nullptr);
    
    param->k_nalloc = 0;
    reallocateDeviceBuffer(&param->log_nk, sits_at->k_numbers, &param->k_numbers, &param->k_nalloc, context);
    copyToDeviceBuffer(&param->log_nk, sits_at->log_nk.data(), 0, sits_at->k_numbers, stream,
                       GpuApiCallBehavior::Async, nullptr);

    /* clear energy and shift force outputs */
    sits_cuda_clear_e_fshift(gpu_sits);
}

gmx_sits_cuda_t* gpu_init_sits(const gmx_device_info_t*   deviceInfo,
                                const sits_atomdata_t*     sits_at,
                                int rank)
{
    hipError_t stat;

    gmx_sits_cuda_t* gpu_sits;
    snew(gpu_sits, 1);
    snew(gpu_sits->sits_atdat, 1);
    snew(gpu_sits->sits_param, 1);
    snew(gpu_sits->stream, 1);


    /* init nbst */
    // pmalloc((void**)&nb->nbst.e_lj, sizeof(*nb->nbst.e_lj));
    // pmalloc((void**)&nb->nbst.e_el, sizeof(*nb->nbst.e_el));
    // pmalloc((void**)&nb->nbst.fshift, SHIFTS * sizeof(*nb->nbst.fshift));

    /* set device info, just point it to the right GPU among the detected ones */
    gpu_sits->dev_info = deviceInfo;

    /* local/non-local GPU streams */
    stat = hipStreamCreate(gpu_sits->stream);
    CU_RET_ERR(stat, "hipStreamCreate on stream failed");

    /* set the kernel type for the current GPU */
    /* pick L1 cache configuration */
    // cuda_set_cacheconfig();

    cuda_init_sits_params(gpu_sits, sits_at);

    gpu_sits->sits_atdat->atomIndicesSize       = 0;
    gpu_sits->sits_atdat->atomIndicesSize_alloc = 0;
    gpu_sits->sits_atdat->sits_cal_mode         = sits_at->sits_cal_mode;        // sits calculation mode: classical or simple
    gpu_sits->sits_atdat->sits_enh_mode         = sits_at->sits_enh_mode; // sits enhancing region: solvate, intramolecular or intermolecular
    gpu_sits->sits_atdat->sits_enh_bias         = sits_at->sits_enh_bias;    // whether to enhance the bias
    gpu_sits->sits_atdat->pw_enh_factor         = sits_at->pw_enh_factor;

    if (debug)
    {
        fprintf(debug, "Initialized SITS CUDA data structures.\n");
    }

    return gpu_sits;
}

/*! Clears the first natoms_clear elements of the GPU nonbonded force output array. */
static void sits_cuda_clear_f(gmx_sits_cuda_t* gpu_sits, int natoms_clear)
{
    hipError_t    stat;
    cu_sits_atdat_t* adat = gpu_sits->sits_atdat;
    hipStream_t   ls   = *(gpu_sits->stream);

    stat = hipMemsetAsync(adat->d_force_tot, 0, natoms_clear * sizeof(*adat->d_force_tot), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on f failed");
    stat = hipMemsetAsync(adat->d_force_pw, 0, natoms_clear * sizeof(*adat->d_force_pw), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on f failed");
    stat = hipMemsetAsync(adat->d_force_tot_nbat, 0, natoms_clear * sizeof(*adat->d_force_tot_nbat), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on f failed");
    stat = hipMemsetAsync(adat->d_force_pw_nbat, 0, natoms_clear * sizeof(*adat->d_force_pw_nbat), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on f failed");
}

/*! Clears nonbonded shift force output array and energy outputs on the GPU. */
static void sits_cuda_clear_e_fshift(gmx_sits_cuda_t* gpu_sits)
{
    hipError_t      stat;
    cu_sits_atdat_t* adat = gpu_sits->sits_atdat;
    hipStream_t     ls   = *(gpu_sits->stream);

    stat = hipMemsetAsync(adat->d_enerd, 0, 3 * sizeof(*adat->d_enerd), ls);
    CU_RET_ERR(stat, "hipMemsetAsync on enerd failed");
    // stat = hipMemsetAsync(adat->e_lj, 0, sizeof(*adat->e_lj), ls);
    // CU_RET_ERR(stat, "hipMemsetAsync on e_lj failed");
    // stat = hipMemsetAsync(adat->e_el, 0, sizeof(*adat->e_el), ls);
    // CU_RET_ERR(stat, "hipMemsetAsync on e_el failed");
}

void sits_gpu_clear_outputs(gmx_sits_cuda_t* gpu_sits, bool computeVirial)
{
    sits_cuda_clear_f(gpu_sits, gpu_sits->sits_atdat->natoms);
    /* clear shift force array and energies if the outputs were
       used in the current step */
    if (true)
    {
        sits_cuda_clear_e_fshift(gpu_sits);
    }
}

void gpu_init_sits_atomdata(gmx_sits_cuda_t* gpu_sits, const nbnxn_atomdata_t* nbat)
{
    hipError_t      stat;
    int              nalloc, natoms;
    bool             realloced;
    cu_sits_atdat_t* d_atdat = gpu_sits->sits_atdat;
    hipStream_t     ls   = *(gpu_sits->stream);

    natoms    = nbat->numAtoms();
    realloced = false;

    if (nbat->params().nenergrp > 1)
    {
        d_atdat->nenergrp = nbat->params().nenergrp;
        d_atdat->neg_2log = nbat->params().neg_2log;
    }

    /* need to reallocate if we have to copy more atoms than the amount of space
       available and only allocate if we haven't initialized yet, i.e d_atdat->natoms == -1 */
    if (natoms > d_atdat->nalloc)
    {
        nalloc = over_alloc_small(natoms);

        /* free up first if the arrays have already been initialized */
        if (d_atdat->nalloc != -1)
        {
            freeDeviceBuffer(&d_atdat->d_force_tot);
            freeDeviceBuffer(&d_atdat->d_force_pw);
            freeDeviceBuffer(&d_atdat->d_force_tot_nbat);
            freeDeviceBuffer(&d_atdat->d_force_pw_nbat);
            freeDeviceBuffer(&d_atdat->energrp);
        }

        stat = hipMalloc((void**)&d_atdat->d_force_tot, nalloc * sizeof(*d_atdat->d_force_tot));
        CU_RET_ERR(stat, "hipMalloc failed on d_atdat->d_force_tot");
        stat = hipMalloc((void**)&d_atdat->d_force_pw, nalloc * sizeof(*d_atdat->d_force_pw));
        CU_RET_ERR(stat, "hipMalloc failed on d_atdat->d_force_pw");
        stat = hipMalloc((void**)&d_atdat->d_force_tot_nbat, nalloc * sizeof(*d_atdat->d_force_tot_nbat));
        CU_RET_ERR(stat, "hipMalloc failed on d_atdat->d_force_tot");
        stat = hipMalloc((void**)&d_atdat->d_force_pw_nbat, nalloc * sizeof(*d_atdat->d_force_pw_nbat));
        CU_RET_ERR(stat, "hipMalloc failed on d_atdat->d_force_pw");
        if (nbat->params().nenergrp > 1)
        {
            stat = hipMalloc((void**)&d_atdat->energrp, nalloc * sizeof(*d_atdat->energrp));
            CU_RET_ERR(stat, "hipMalloc failed on d_atdat->energrp");
        }

        d_atdat->nalloc = nalloc;
        realloced       = true;
    }

    d_atdat->natoms       = natoms;

    /* need to clear GPU f output if realloc happened */
    if (realloced)
    {
        sits_cuda_clear_f(gpu_sits, nalloc);
    }

    if (nbat->params().nenergrp > 1)
    {
        cu_copy_H2D_async(d_atdat->energrp, nbat->params().energrp_1x1.data(),
                          natoms * sizeof(*d_atdat->energrp), ls);
    }
}

void gpu_print_sitsvals(gmx_sits_cuda_t* gpu_sits)
{
    float* h_enerd;
    h_enerd = (float *) malloc(3 * sizeof(float));
    hipMemcpy(h_enerd, gpu_sits->sits_atdat->d_enerd, 3*sizeof(float), hipMemcpyDeviceToHost);

    float* h_factor;
    h_factor = (float *) malloc(sizeof(float));
    hipMemcpy(h_factor, gpu_sits->sits_param->factor, sizeof(float), hipMemcpyDeviceToHost);

    // float* h_sum_a;
    // h_sum_a = (float *) malloc(sizeof(float));
    // hipMemcpy(h_sum_a, gpu_sits->sits_param->sum_a, sizeof(float), hipMemcpyDeviceToHost);

    // float* h_sum_b;
    // h_sum_a = (float *) malloc(sizeof(float));
    // hipMemcpy(h_sum_a, gpu_sits->sits_param->sum_a, sizeof(float), hipMemcpyDeviceToHost);

    printf("\n______AA______ ______AB______ ______BB______   sum_a  sum_b  fc_ball\n");
    printf("%14.4f %14.4f %14.4f %7.4f\n", h_enerd[0], h_enerd[1], h_enerd[2], h_factor[0]);
}

void gpu_free(gmx_sits_cuda_t* gpu_sits)
{
    hipError_t      stat;
    cu_sits_atdat_t* atdat;
    cu_sits_param_t* param;

    if (gpu_sits == nullptr)
    {
        return;
    }

    atdat = gpu_sits->sits_atdat;
    param = gpu_sits->sits_param;

    // if ((info.sits_mode & 0x0000000F) == SIMPLE_SITS_MODE)
    // {
    //     if (simple_info.fc_pdf != NULL)
    //     {
    //         free(simple_info.fc_pdf);
    //     }
    // }

    if (atdat->d_enerd != nullptr)
    {
        // TODO: fix hipFree here!
        // stat = hipFree(atdat->d_enerd);
        // CU_RET_ERR(stat, "hipFree failed on atdat->d_enerd");
    }

    // freeDeviceBuffer(&atdat->d_force_tot);
    // freeDeviceBuffer(&atdat->d_force_pw);
    // freeDeviceBuffer(&atdat->d_force_tot_nbat);
    // freeDeviceBuffer(&atdat->d_force_pw_nbat);
    // freeDeviceBuffer(&atdat->atomIndices);
    // freeDeviceBuffer(&atdat->energrp);

    /* Free nbst */
    // pfree(nb->nbst.e_lj);
    // nb->nbst.e_lj = nullptr;

    // pfree(nb->nbst.e_el);
    // nb->nbst.e_el = nullptr;

    // pfree(nb->nbst.fshift);
    // nb->nbst.fshift = nullptr;

    sfree(atdat);
    sfree(param);
    sfree(gpu_sits->stream);
    sfree(gpu_sits);

    if (debug)
    {
        fprintf(debug, "Cleaned up CUDA data structures.\n");
    }
}

// void* gpu_get_xq(gmx_nbnxn_gpu_t* nb)
// {
//     assert(nb);

//     return static_cast<void*>(nb->atdat->xq);
// }

// void* gpu_get_f(gmx_nbnxn_gpu_t* nb)
// {
//     assert(nb);

//     return static_cast<void*>(nb->atdat->f);
// }

} // namespace Sits