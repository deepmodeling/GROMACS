#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2018,2019, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \internal \file
 *
 * \brief Implements CUDA bonded functionality
 *
 * \author Jon Vincent <jvincent@nvidia.com>
 * \author Magnus Lundborg <lundborg.magnus@gmail.com>
 * \author Berk Hess <hess@kth.se>
 * \author Szilárd Páll <pall.szilard@gmail.com>
 * \author Alan Gray <alang@nvidia.com>
 * \author Mark Abraham <mark.j.abraham@gmail.com>
 *
 * \ingroup module_listed_forces
 */

#include "gmxpre.h"

#include <cassert>

#include <hip/hip_math_constants.h>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/gpu_vec.cuh"
#include "gromacs/listed_forces/gpubonded.h"
#include "gromacs/math/units.h"
#include "gromacs/mdlib/force_flags.h"
#include "gromacs/mdtypes/forcerec.h"
#include "gromacs/mdtypes/simulation_workload.h"
#include "gromacs/pbcutil/pbc.h"
#include "gromacs/pbcutil/pbc_aiuc_cuda.cuh"
#include "gromacs/utility/gmxassert.h"

#include "gpubonded_impl.h"

struct BondedFepParameters;

#if defined(_MSVC)
#    include <limits>
#endif

// CUDA threads per block
#define TPB_BONDED 256

/*-------------------------------- CUDA kernels-------------------------------- */
/*------------------------------------------------------------------------------*/

#define CUDA_DEG2RAD_F (HIP_PI_F / 180.0f)

/*---------------- BONDED CUDA kernels--------------*/

/* Harmonic */
__device__ __forceinline__ static void
           harmonic_gpu(const float kA, const float xA, const float x, float* V, float* F)
{
    constexpr float half = 0.5f;
    float           dx, dx2;

    dx  = x - xA;
    dx2 = dx * dx;

    *F = -kA * dx;
    *V = half * kA * dx2;
}

__device__ __forceinline__ static void
           harmonic_fep_gpu(const float kA, const float kB, const float xA, const float xB, const float x, const float lambda, float* V, float* F)
{
    constexpr float half = 0.5f;
    float           L1, kk, x0, dx, dx2;
    float           v, f, dvdlambda;

    L1 = 1.0 - lambda;
    kk = L1 * kA + lambda * kB;
    x0 = L1 * xA + lambda * xB;

    dx  = x - x0;
    dx2 = dx * dx;

    f         = -kk * dx;
    v         = half * kk * dx2;
    dvdlambda = half * (kB - kA) * dx2 + (xA - xB) * kk * dx;

    *F = f;
    *V = v;
}

__device__ __forceinline__ static void
           harmonic_softbond_gpu(const float kA, const float kB, const float xA, const float xB, const float x, const float sb_alpha, const float lambda, float* V, float* F)
{
    constexpr float half = 0.5f;
    float           L1, kk, x0, dx, dx2, dxA, dxA2, dxB, dxB2;
    float           v, f, dvdlambda;

    L1 = 1.0 - lambda;
    kk = L1 * kA + lambda * kB;
    x0 = L1 * xA + lambda * xB;

    dx  = x - x0;
    dx2 = dx * dx;

    dxA = x - xA;
    dxA2= dxA * dxA;

    dxB = x - xB;
    dxB2= dxB * dxB;

    if (kA == 0)
    {
        f         = -lambda * kB * dxB / (1 + sb_alpha * L1 * dxB2) / (1 + sb_alpha * L1 * dxB2);
        v         = half * lambda * kB * dxB2 / (1 + sb_alpha * L1 * dxB2);
        dvdlambda = v * (1 + sb_alpha * dxB2) / (1 + sb_alpha * L1 * dxB2);
    }
    else if (kB == 0)
    {
        f         = -L1 * kA * dxA / (1 + sb_alpha * lambda * dxB2) / (1 + sb_alpha * lambda * dxB2);
        v         = half * L1 * kA * dxA2 / (1 + sb_alpha * lambda * dxA2);
        dvdlambda = v * (1 + sb_alpha * dxB2) / (1 + sb_alpha * lambda * dxB2);
    }
    else
    {
        f         = -kk * dx;
        v         = half * kk * dx2;
        dvdlambda = half * (kB - kA) * dx2 + (xA - xB) * kk * dx;
    }

    *F = f;
    *V = v;
}

template<bool calcVir, bool calcEner>
__device__ void bonds_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          const float4    gm_xq[],
                          fvec            gm_f[],
                          fvec            sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int3 bondData = *(int3*)(d_forceatoms + 3 * i);
        int  type     = bondData.x;
        int  ai       = bondData.y;
        int  aj       = bondData.z;

        /* dx = xi - xj, corrected for periodic boundary conditions. */
        fvec dx;
        int  ki = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[aj], dx);

        float dr2 = iprod_gpu(dx, dx);
        float dr  = sqrt(dr2);

        float vbond;
        float fbond;
        harmonic_gpu(d_forceparams[type].harmonic.krA, d_forceparams[type].harmonic.rA, dr, &vbond, &fbond);

        if (calcEner)
        {
            *vtot_loc += vbond;
        }

        if (dr2 != 0.0f)
        {
            fbond *= rsqrtf(dr2);

#pragma unroll
            for (int m = 0; m < DIM; m++)
            {
                float fij = fbond * dx[m];
                atomicAdd(&gm_f[ai][m], fij);
                atomicAdd(&gm_f[aj][m], -fij);
                if (calcVir && ki != CENTRAL)
                {
                    atomicAdd(&sm_fShiftLoc[ki][m], fij);
                    atomicAdd(&sm_fShiftLoc[CENTRAL][m], -fij);
                }
            }
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void bonds_fep_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          gmx::BondedFepParameters* d_fepparams,
                          const float4    gm_xq[],
                          fvec            gm_f[],
                          fvec            sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int3 bondData = *(int3*)(d_forceatoms + 3 * i);
        int  type     = bondData.x;
        int  ai       = bondData.y;
        int  aj       = bondData.z;

        /* dx = xi - xj, corrected for periodic boundary conditions. */
        fvec dx;
        int  ki = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[aj], dx);

        float dr2 = iprod_gpu(dx, dx);
        float dr  = sqrt(dr2);

        float vbond;
        float fbond;
        harmonic_softbond_gpu(d_forceparams[type].harmonic.krA, d_forceparams[type].harmonic.krB, d_forceparams[type].harmonic.rA, d_forceparams[type].harmonic.rB, 
                              dr, d_fepparams->alpha_bond, d_fepparams->lambda_q, &vbond, &fbond);

        if (calcEner)
        {
            *vtot_loc += vbond;
        }

        if (dr2 != 0.0f)
        {
            fbond *= rsqrtf(dr2);

#pragma unroll
            for (int m = 0; m < DIM; m++)
            {
                float fij = fbond * dx[m];
                atomicAdd(&gm_f[ai][m], fij);
                atomicAdd(&gm_f[aj][m], -fij);
                if (calcVir && ki != CENTRAL)
                {
                    atomicAdd(&sm_fShiftLoc[ki][m], fij);
                    atomicAdd(&sm_fShiftLoc[CENTRAL][m], -fij);
                }
            }
        }
    }
}

template<bool returnShift>
__device__ __forceinline__ static float bond_angle_gpu(const float4   xi,
                                                       const float4   xj,
                                                       const float4   xk,
                                                       const PbcAiuc& pbcAiuc,
                                                       fvec           r_ij,
                                                       fvec           r_kj,
                                                       float*         costh,
                                                       int*           t1,
                                                       int*           t2)
/* Return value is the angle between the bonds i-j and j-k */
{
    *t1 = pbcDxAiuc<returnShift>(pbcAiuc, xi, xj, r_ij);
    *t2 = pbcDxAiuc<returnShift>(pbcAiuc, xk, xj, r_kj);

    *costh   = cos_angle_gpu(r_ij, r_kj);
    float th = acosf(*costh);

    return th;
}

template<bool calcVir, bool calcEner>
__device__ void angles_gpu(const int       i,
                           float*          vtot_loc,
                           const int       numBonds,
                           const t_iatom   d_forceatoms[],
                           const t_iparams d_forceparams[],
                           const float4    gm_xq[],
                           fvec            gm_f[],
                           fvec            sm_fShiftLoc[],
                           const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int4 angleData = *(int4*)(d_forceatoms + 4 * i);
        int  type      = angleData.x;
        int  ai        = angleData.y;
        int  aj        = angleData.z;
        int  ak        = angleData.w;

        fvec  r_ij;
        fvec  r_kj;
        float cos_theta;
        int   t1;
        int   t2;
        float theta = bond_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], pbcAiuc, r_ij, r_kj,
                                              &cos_theta, &t1, &t2);

        float va;
        float dVdt;
        harmonic_gpu(d_forceparams[type].harmonic.krA,
                     d_forceparams[type].harmonic.rA * CUDA_DEG2RAD_F, theta, &va, &dVdt);

        if (calcEner)
        {
            *vtot_loc += va;
        }

        float cos_theta2 = cos_theta * cos_theta;
        if (cos_theta2 < 1.0f)
        {
            float st    = dVdt * rsqrtf(1.0f - cos_theta2);
            float sth   = st * cos_theta;
            float nrij2 = iprod_gpu(r_ij, r_ij);
            float nrkj2 = iprod_gpu(r_kj, r_kj);

            float nrij_1 = rsqrtf(nrij2);
            float nrkj_1 = rsqrtf(nrkj2);

            float cik = st * nrij_1 * nrkj_1;
            float cii = sth * nrij_1 * nrij_1;
            float ckk = sth * nrkj_1 * nrkj_1;

            fvec f_i;
            fvec f_k;
            fvec f_j;
#pragma unroll
            for (int m = 0; m < DIM; m++)
            {
                f_i[m] = -(cik * r_kj[m] - cii * r_ij[m]);
                f_k[m] = -(cik * r_ij[m] - ckk * r_kj[m]);
                f_j[m] = -f_i[m] - f_k[m];
                atomicAdd(&gm_f[ai][m], f_i[m]);
                atomicAdd(&gm_f[aj][m], f_j[m]);
                atomicAdd(&gm_f[ak][m], f_k[m]);
                if (calcVir)
                {
                    atomicAdd(&sm_fShiftLoc[t1][m], f_i[m]);
                    atomicAdd(&sm_fShiftLoc[CENTRAL][m], f_j[m]);
                    atomicAdd(&sm_fShiftLoc[t2][m], f_k[m]);
                }
            }
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void angles_fep_gpu(const int       i,
                           float*          vtot_loc,
                           const int       numBonds,
                           const t_iatom   d_forceatoms[],
                           const t_iparams d_forceparams[],
                           gmx::BondedFepParameters* d_fepparams,
                           const float4    gm_xq[],
                           fvec            gm_f[],
                           fvec            sm_fShiftLoc[],
                           const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int4 angleData = *(int4*)(d_forceatoms + 4 * i);
        int  type      = angleData.x;
        int  ai        = angleData.y;
        int  aj        = angleData.z;
        int  ak        = angleData.w;

        fvec  r_ij;
        fvec  r_kj;
        float cos_theta;
        int   t1;
        int   t2;
        float theta = bond_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], pbcAiuc, r_ij, r_kj,
                                              &cos_theta, &t1, &t2);

        float va;
        float dVdt;
        harmonic_fep_gpu(d_forceparams[type].harmonic.krA, d_forceparams[type].harmonic.krB,
                     d_forceparams[type].harmonic.rA * CUDA_DEG2RAD_F, d_forceparams[type].harmonic.rB * CUDA_DEG2RAD_F, theta, d_fepparams->lambda_q, &va, &dVdt);

        if (calcEner)
        {
            *vtot_loc += va;
        }

        float cos_theta2 = cos_theta * cos_theta;
        if (cos_theta2 < 1.0f)
        {
            float st    = dVdt * rsqrtf(1.0f - cos_theta2);
            float sth   = st * cos_theta;
            float nrij2 = iprod_gpu(r_ij, r_ij);
            float nrkj2 = iprod_gpu(r_kj, r_kj);

            float nrij_1 = rsqrtf(nrij2);
            float nrkj_1 = rsqrtf(nrkj2);

            float cik = st * nrij_1 * nrkj_1;
            float cii = sth * nrij_1 * nrij_1;
            float ckk = sth * nrkj_1 * nrkj_1;

            fvec f_i;
            fvec f_k;
            fvec f_j;
#pragma unroll
            for (int m = 0; m < DIM; m++)
            {
                f_i[m] = -(cik * r_kj[m] - cii * r_ij[m]);
                f_k[m] = -(cik * r_ij[m] - ckk * r_kj[m]);
                f_j[m] = -f_i[m] - f_k[m];
                atomicAdd(&gm_f[ai][m], f_i[m]);
                atomicAdd(&gm_f[aj][m], f_j[m]);
                atomicAdd(&gm_f[ak][m], f_k[m]);
                if (calcVir)
                {
                    atomicAdd(&sm_fShiftLoc[t1][m], f_i[m]);
                    atomicAdd(&sm_fShiftLoc[CENTRAL][m], f_j[m]);
                    atomicAdd(&sm_fShiftLoc[t2][m], f_k[m]);
                }
            }
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void urey_bradley_gpu(const int       i,
                                 float*          vtot_loc,
                                 const int       numBonds,
                                 const t_iatom   d_forceatoms[],
                                 const t_iparams d_forceparams[],
                                 const float4    gm_xq[],
                                 fvec            gm_f[],
                                 fvec            sm_fShiftLoc[],
                                 const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int4 ubData = *(int4*)(d_forceatoms + 4 * i);
        int  type   = ubData.x;
        int  ai     = ubData.y;
        int  aj     = ubData.z;
        int  ak     = ubData.w;

        float th0A = d_forceparams[type].u_b.thetaA * CUDA_DEG2RAD_F;
        float kthA = d_forceparams[type].u_b.kthetaA;
        float r13A = d_forceparams[type].u_b.r13A;
        float kUBA = d_forceparams[type].u_b.kUBA;

        fvec  r_ij;
        fvec  r_kj;
        float cos_theta;
        int   t1;
        int   t2;
        float theta = bond_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], pbcAiuc, r_ij, r_kj,
                                              &cos_theta, &t1, &t2);

        float va;
        float dVdt;
        harmonic_gpu(kthA, th0A, theta, &va, &dVdt);

        if (calcEner)
        {
            *vtot_loc += va;
        }

        fvec r_ik;
        int  ki = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[ak], r_ik);

        float dr2 = iprod_gpu(r_ik, r_ik);
        float dr  = dr2 * rsqrtf(dr2);

        float vbond;
        float fbond;
        harmonic_gpu(kUBA, r13A, dr, &vbond, &fbond);

        float cos_theta2 = cos_theta * cos_theta;
        if (cos_theta2 < 1.0f)
        {
            float st  = dVdt * rsqrtf(1.0f - cos_theta2);
            float sth = st * cos_theta;

            float nrkj2 = iprod_gpu(r_kj, r_kj);
            float nrij2 = iprod_gpu(r_ij, r_ij);

            float cik = st * rsqrtf(nrkj2 * nrij2);
            float cii = sth / nrij2;
            float ckk = sth / nrkj2;

            fvec f_i;
            fvec f_j;
            fvec f_k;
#pragma unroll
            for (int m = 0; m < DIM; m++)
            {
                f_i[m] = -(cik * r_kj[m] - cii * r_ij[m]);
                f_k[m] = -(cik * r_ij[m] - ckk * r_kj[m]);
                f_j[m] = -f_i[m] - f_k[m];
                atomicAdd(&gm_f[ai][m], f_i[m]);
                atomicAdd(&gm_f[aj][m], f_j[m]);
                atomicAdd(&gm_f[ak][m], f_k[m]);
                if (calcVir)
                {
                    atomicAdd(&sm_fShiftLoc[t1][m], f_i[m]);
                    atomicAdd(&sm_fShiftLoc[CENTRAL][m], f_j[m]);
                    atomicAdd(&sm_fShiftLoc[t2][m], f_k[m]);
                }
            }
        }

        /* Time for the bond calculations */
        if (dr2 != 0.0f)
        {
            if (calcEner)
            {
                *vtot_loc += vbond;
            }

            fbond *= rsqrtf(dr2);

#pragma unroll
            for (int m = 0; m < DIM; m++)
            {
                float fik = fbond * r_ik[m];
                atomicAdd(&gm_f[ai][m], fik);
                atomicAdd(&gm_f[ak][m], -fik);

                if (calcVir && ki != CENTRAL)
                {
                    atomicAdd(&sm_fShiftLoc[ki][m], fik);
                    atomicAdd(&sm_fShiftLoc[CENTRAL][m], -fik);
                }
            }
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void urey_bradley_fep_gpu(const int       i,
                                 float*          vtot_loc,
                                 const int       numBonds,
                                 const t_iatom   d_forceatoms[],
                                 const t_iparams d_forceparams[],
                                 gmx::BondedFepParameters* d_fepparams,
                                 const float4    gm_xq[],
                                 fvec            gm_f[],
                                 fvec            sm_fShiftLoc[],
                                 const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int4 ubData = *(int4*)(d_forceatoms + 4 * i);
        int  type   = ubData.x;
        int  ai     = ubData.y;
        int  aj     = ubData.z;
        int  ak     = ubData.w;

        float th0A = d_forceparams[type].u_b.thetaA * CUDA_DEG2RAD_F;
        float kthA = d_forceparams[type].u_b.kthetaA;
        float r13A = d_forceparams[type].u_b.r13A;
        float kUBA = d_forceparams[type].u_b.kUBA;

        float th0B = d_forceparams[type].u_b.thetaB * CUDA_DEG2RAD_F;
        float kthB = d_forceparams[type].u_b.kthetaB;
        float r13B = d_forceparams[type].u_b.r13B;
        float kUBB = d_forceparams[type].u_b.kUBB;

        fvec  r_ij;
        fvec  r_kj;
        float cos_theta;
        int   t1;
        int   t2;
        float theta = bond_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], pbcAiuc, r_ij, r_kj,
                                              &cos_theta, &t1, &t2);

        float va;
        float dVdt;
        // harmonic_gpu(kthA, th0A, theta, &va, &dVdt);
        harmonic_fep_gpu(kthA, kthB, th0A, th0B, theta, d_fepparams->lambda_q, &va, &dVdt);

        if (calcEner)
        {
            *vtot_loc += va;
        }

        fvec r_ik;
        int  ki = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[ak], r_ik);

        float dr2 = iprod_gpu(r_ik, r_ik);
        float dr  = dr2 * rsqrtf(dr2);

        float vbond;
        float fbond;
        // harmonic_gpu(kUBA, r13A, dr, &vbond, &fbond);
        harmonic_fep_gpu(kUBA, kUBB, r13A, r13B, dr, d_fepparams->lambda_q, &vbond, &fbond);

        float cos_theta2 = cos_theta * cos_theta;
        if (cos_theta2 < 1.0f)
        {
            float st  = dVdt * rsqrtf(1.0f - cos_theta2);
            float sth = st * cos_theta;

            float nrkj2 = iprod_gpu(r_kj, r_kj);
            float nrij2 = iprod_gpu(r_ij, r_ij);

            float cik = st * rsqrtf(nrkj2 * nrij2);
            float cii = sth / nrij2;
            float ckk = sth / nrkj2;

            fvec f_i;
            fvec f_j;
            fvec f_k;
#pragma unroll
            for (int m = 0; m < DIM; m++)
            {
                f_i[m] = -(cik * r_kj[m] - cii * r_ij[m]);
                f_k[m] = -(cik * r_ij[m] - ckk * r_kj[m]);
                f_j[m] = -f_i[m] - f_k[m];
                atomicAdd(&gm_f[ai][m], f_i[m]);
                atomicAdd(&gm_f[aj][m], f_j[m]);
                atomicAdd(&gm_f[ak][m], f_k[m]);
                if (calcVir)
                {
                    atomicAdd(&sm_fShiftLoc[t1][m], f_i[m]);
                    atomicAdd(&sm_fShiftLoc[CENTRAL][m], f_j[m]);
                    atomicAdd(&sm_fShiftLoc[t2][m], f_k[m]);
                }
            }
        }

        /* Time for the bond calculations */
        if (dr2 != 0.0f)
        {
            if (calcEner)
            {
                *vtot_loc += vbond;
            }

            fbond *= rsqrtf(dr2);

#pragma unroll
            for (int m = 0; m < DIM; m++)
            {
                float fik = fbond * r_ik[m];
                atomicAdd(&gm_f[ai][m], fik);
                atomicAdd(&gm_f[ak][m], -fik);

                if (calcVir && ki != CENTRAL)
                {
                    atomicAdd(&sm_fShiftLoc[ki][m], fik);
                    atomicAdd(&sm_fShiftLoc[CENTRAL][m], -fik);
                }
            }
        }
    }
}

template<bool returnShift, typename T>
__device__ __forceinline__ static float dih_angle_gpu(const T        xi,
                                                      const T        xj,
                                                      const T        xk,
                                                      const T        xl,
                                                      const PbcAiuc& pbcAiuc,
                                                      fvec           r_ij,
                                                      fvec           r_kj,
                                                      fvec           r_kl,
                                                      fvec           m,
                                                      fvec           n,
                                                      int*           t1,
                                                      int*           t2,
                                                      int*           t3)
{
    *t1 = pbcDxAiuc<returnShift>(pbcAiuc, xi, xj, r_ij);
    *t2 = pbcDxAiuc<returnShift>(pbcAiuc, xk, xj, r_kj);
    *t3 = pbcDxAiuc<returnShift>(pbcAiuc, xk, xl, r_kl);

    cprod_gpu(r_ij, r_kj, m);
    cprod_gpu(r_kj, r_kl, n);
    float phi  = gmx_angle_gpu(m, n);
    float ipr  = iprod_gpu(r_ij, n);
    float sign = (ipr < 0.0f) ? -1.0f : 1.0f;
    phi        = sign * phi;

    return phi;
}


__device__ __forceinline__ static void
           dopdihs_gpu(const float cpA, const float phiA, const int mult, const float phi, float* v, float* f)
{
    float mdphi, sdphi;

    mdphi = mult * phi - phiA * CUDA_DEG2RAD_F;
    sdphi = sinf(mdphi);
    *v    = cpA * (1.0f + cosf(mdphi));
    *f    = -cpA * mult * sdphi;
}

__device__ __forceinline__ static void
           dopdihs_fep_gpu(const float cpA, const float cpB, const float phiA, const float phiB, const int mult, const float phi, const float lambda, float* v, float* f)
{
    float mdphiA, sdphiA, mdphiB, sdphiB;
    mdphiA = mult * phi - phiA * CUDA_DEG2RAD_F;
    mdphiB = mult * phi - phiB * CUDA_DEG2RAD_F;
    sdphiA = sinf(mdphiA);
    sdphiB = sinf(mdphiB);
    *v    = (1 - lambda) * cpA * (1.0f + cosf(mdphiA)) + lambda * cpB * (1.0f + cosf(mdphiB));
    *f    = -((1 - lambda) * cpA * mult * sdphiA + lambda * cpB * mult * sdphiB);
}

template<bool calcVir>
__device__ static void do_dih_fup_gpu(const int      i,
                                      const int      j,
                                      const int      k,
                                      const int      l,
                                      const float    ddphi,
                                      const fvec     r_ij,
                                      const fvec     r_kj,
                                      const fvec     r_kl,
                                      const fvec     m,
                                      const fvec     n,
                                      fvec           gm_f[],
                                      fvec           sm_fShiftLoc[],
                                      const PbcAiuc& pbcAiuc,
                                      const float4   gm_xq[],
                                      const int      t1,
                                      const int      t2,
                                      const int gmx_unused t3)
{
    float iprm  = iprod_gpu(m, m);
    float iprn  = iprod_gpu(n, n);
    float nrkj2 = iprod_gpu(r_kj, r_kj);
    float toler = nrkj2 * GMX_REAL_EPS;
    if ((iprm > toler) && (iprn > toler))
    {
        float nrkj_1 = rsqrtf(nrkj2); // replacing std::invsqrt call
        float nrkj_2 = nrkj_1 * nrkj_1;
        float nrkj   = nrkj2 * nrkj_1;
        float a      = -ddphi * nrkj / iprm;
        fvec  f_i;
        svmul_gpu(a, m, f_i);
        float b = ddphi * nrkj / iprn;
        fvec  f_l;
        svmul_gpu(b, n, f_l);
        float p = iprod_gpu(r_ij, r_kj);
        p *= nrkj_2;
        float q = iprod_gpu(r_kl, r_kj);
        q *= nrkj_2;
        fvec uvec;
        svmul_gpu(p, f_i, uvec);
        fvec vvec;
        svmul_gpu(q, f_l, vvec);
        fvec svec;
        fvec_sub_gpu(uvec, vvec, svec);
        fvec f_j;
        fvec_sub_gpu(f_i, svec, f_j);
        fvec f_k;
        fvec_add_gpu(f_l, svec, f_k);
#pragma unroll
        for (int m = 0; (m < DIM); m++)
        {
            atomicAdd(&gm_f[i][m], f_i[m]);
            atomicAdd(&gm_f[j][m], -f_j[m]);
            atomicAdd(&gm_f[k][m], -f_k[m]);
            atomicAdd(&gm_f[l][m], f_l[m]);
        }

        if (calcVir)
        {
            fvec dx_jl;
            int  t3 = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[l], gm_xq[j], dx_jl);

#pragma unroll
            for (int m = 0; (m < DIM); m++)
            {
                atomicAdd(&sm_fShiftLoc[t1][m], f_i[m]);
                atomicAdd(&sm_fShiftLoc[CENTRAL][m], -f_j[m]);
                atomicAdd(&sm_fShiftLoc[t2][m], -f_k[m]);
                atomicAdd(&sm_fShiftLoc[t3][m], f_l[m]);
            }
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void pdihs_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          const float4    gm_xq[],
                          fvec            gm_f[],
                          fvec            sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        fvec  r_ij;
        fvec  r_kj;
        fvec  r_kl;
        fvec  m;
        fvec  n;
        int   t1;
        int   t2;
        int   t3;
        float phi = dih_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc,
                                           r_ij, r_kj, r_kl, m, n, &t1, &t2, &t3);

        float vpd;
        float ddphi;
        dopdihs_gpu(d_forceparams[type].pdihs.cpA, d_forceparams[type].pdihs.phiA,
                    d_forceparams[type].pdihs.mult, phi, &vpd, &ddphi);

        if (calcEner)
        {
            *vtot_loc += vpd;
        }

        do_dih_fup_gpu<calcVir>(ai, aj, ak, al, ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc,
                                pbcAiuc, gm_xq, t1, t2, t3);
    }
}

template<bool calcVir, bool calcEner>
__device__ void pdihs_fep_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          gmx::BondedFepParameters* d_fepparams,
                          const float4    gm_xq[],
                          fvec            gm_f[],
                          fvec            sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        fvec  r_ij;
        fvec  r_kj;
        fvec  r_kl;
        fvec  m;
        fvec  n;
        int   t1;
        int   t2;
        int   t3;
        float phi = dih_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc,
                                           r_ij, r_kj, r_kl, m, n, &t1, &t2, &t3);

        float vpd;
        float ddphi;
        dopdihs_fep_gpu(d_forceparams[type].pdihs.cpA, d_forceparams[type].pdihs.cpB, d_forceparams[type].pdihs.phiA, d_forceparams[type].pdihs.phiB,
                    d_forceparams[type].pdihs.mult, phi, d_fepparams->lambda_q, &vpd, &ddphi);

        if (calcEner)
        {
            *vtot_loc += vpd;
        }

        do_dih_fup_gpu<calcVir>(ai, aj, ak, al, ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc,
                                pbcAiuc, gm_xq, t1, t2, t3);
    }
}

template<bool calcVir, bool calcEner>
__device__ void rbdihs_gpu(const int       i,
                           float*          vtot_loc,
                           const int       numBonds,
                           const t_iatom   d_forceatoms[],
                           const t_iparams d_forceparams[],
                           const float4    gm_xq[],
                           fvec            gm_f[],
                           fvec            sm_fShiftLoc[],
                           const PbcAiuc   pbcAiuc)
{
    constexpr float c0 = 0.0f, c1 = 1.0f, c2 = 2.0f, c3 = 3.0f, c4 = 4.0f, c5 = 5.0f;

    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        fvec  r_ij;
        fvec  r_kj;
        fvec  r_kl;
        fvec  m;
        fvec  n;
        int   t1;
        int   t2;
        int   t3;
        float phi = dih_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc,
                                           r_ij, r_kj, r_kl, m, n, &t1, &t2, &t3);

        /* Change to polymer convention */
        if (phi < c0)
        {
            phi += HIP_PI_F;
        }
        else
        {
            phi -= HIP_PI_F;
        }
        float cos_phi = cosf(phi);
        /* Beware of accuracy loss, cannot use 1-sqrt(cos^2) ! */
        float sin_phi = sinf(phi);

        float parm[NR_RBDIHS];
        for (int j = 0; j < NR_RBDIHS; j++)
        {
            parm[j] = d_forceparams[type].rbdihs.rbcA[j];
        }
        /* Calculate cosine powers */
        /* Calculate the energy */
        /* Calculate the derivative */
        float v      = parm[0];
        float ddphi  = c0;
        float cosfac = c1;

        float rbp = parm[1];
        ddphi += rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[2];
        ddphi += c2 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[3];
        ddphi += c3 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[4];
        ddphi += c4 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[5];
        ddphi += c5 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }

        ddphi = -ddphi * sin_phi;

        do_dih_fup_gpu<calcVir>(ai, aj, ak, al, ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc,
                                pbcAiuc, gm_xq, t1, t2, t3);
        if (calcEner)
        {
            *vtot_loc += v;
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void rbdihs_fep_gpu(const int       i,
                           float*          vtot_loc,
                           const int       numBonds,
                           const t_iatom   d_forceatoms[],
                           const t_iparams d_forceparams[],
                           gmx::BondedFepParameters* d_fepparams,
                           const float4    gm_xq[],
                           fvec            gm_f[],
                           fvec            sm_fShiftLoc[],
                           const PbcAiuc   pbcAiuc)
{
    constexpr float c0 = 0.0f, c1 = 1.0f, c2 = 2.0f, c3 = 3.0f, c4 = 4.0f, c5 = 5.0f;

    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        fvec  r_ij;
        fvec  r_kj;
        fvec  r_kl;
        fvec  m;
        fvec  n;
        int   t1;
        int   t2;
        int   t3;
        float phi = dih_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc,
                                           r_ij, r_kj, r_kl, m, n, &t1, &t2, &t3);
        float lambda = d_fepparams->lambda_q;

        /* Change to polymer convention */
        if (phi < c0)
        {
            phi += HIP_PI_F;
        }
        else
        {
            phi -= HIP_PI_F;
        }
        float cos_phi = cosf(phi);
        /* Beware of accuracy loss, cannot use 1-sqrt(cos^2) ! */
        float sin_phi = sinf(phi);

        float parm[NR_RBDIHS];
        for (int j = 0; j < NR_RBDIHS; j++)
        {
            parm[j] = (1 - lambda) * d_forceparams[type].rbdihs.rbcA[j] + lambda * d_forceparams[type].rbdihs.rbcB[j];
        }
        /* Calculate cosine powers */
        /* Calculate the energy */
        /* Calculate the derivative */
        float v      = parm[0];
        float ddphi  = c0;
        float cosfac = c1;

        float rbp = parm[1];
        ddphi += rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[2];
        ddphi += c2 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[3];
        ddphi += c3 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[4];
        ddphi += c4 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[5];
        ddphi += c5 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }

        ddphi = -ddphi * sin_phi;

        do_dih_fup_gpu<calcVir>(ai, aj, ak, al, ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc,
                                pbcAiuc, gm_xq, t1, t2, t3);
        if (calcEner)
        {
            *vtot_loc += v;
        }
    }
}

__device__ __forceinline__ static void make_dp_periodic_gpu(float* dp)
{
    /* dp cannot be outside (-pi,pi) */
    if (*dp >= HIP_PI_F)
    {
        *dp -= 2.0f * HIP_PI_F;
    }
    else if (*dp < -HIP_PI_F)
    {
        *dp += 2.0f * HIP_PI_F;
    }
}

template<bool calcVir, bool calcEner>
__device__ void idihs_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          const float4    gm_xq[],
                          fvec            gm_f[],
                          fvec            sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        fvec  r_ij;
        fvec  r_kj;
        fvec  r_kl;
        fvec  m;
        fvec  n;
        int   t1;
        int   t2;
        int   t3;
        float phi = dih_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc,
                                           r_ij, r_kj, r_kl, m, n, &t1, &t2, &t3);

        /* phi can jump if phi0 is close to Pi/-Pi, which will cause huge
         * force changes if we just apply a normal harmonic.
         * Instead, we first calculate phi-phi0 and take it modulo (-Pi,Pi).
         * This means we will never have the periodicity problem, unless
         * the dihedral is Pi away from phiO, which is very unlikely due to
         * the potential.
         */
        float kA = d_forceparams[type].harmonic.krA;
        float pA = d_forceparams[type].harmonic.rA;

        float phi0 = pA * CUDA_DEG2RAD_F;

        float dp = phi - phi0;

        make_dp_periodic_gpu(&dp);

        float ddphi = -kA * dp;

        do_dih_fup_gpu<calcVir>(ai, aj, ak, al, -ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc,
                                pbcAiuc, gm_xq, t1, t2, t3);

        if (calcEner)
        {
            *vtot_loc += -0.5f * ddphi * dp;
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void idihs_fep_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          gmx::BondedFepParameters* d_fepparams,
                          const float4    gm_xq[],
                          fvec            gm_f[],
                          fvec            sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        fvec  r_ij;
        fvec  r_kj;
        fvec  r_kl;
        fvec  m;
        fvec  n;
        int   t1;
        int   t2;
        int   t3;
        float phi = dih_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc,
                                           r_ij, r_kj, r_kl, m, n, &t1, &t2, &t3);

        /* phi can jump if phi0 is close to Pi/-Pi, which will cause huge
         * force changes if we just apply a normal harmonic.
         * Instead, we first calculate phi-phi0 and take it modulo (-Pi,Pi).
         * This means we will never have the periodicity problem, unless
         * the dihedral is Pi away from phiO, which is very unlikely due to
         * the potential.
         */
        float kA = d_forceparams[type].harmonic.krA;
        float pA = d_forceparams[type].harmonic.rA;
        float kB = d_forceparams[type].harmonic.krB;
        float pB = d_forceparams[type].harmonic.rB;

        float phi0A = pA * CUDA_DEG2RAD_F;
        float phi0B = pB * CUDA_DEG2RAD_F;

        float dpA = phi - phi0A;
        float dpB = phi - phi0B;

        make_dp_periodic_gpu(&dpA);
        make_dp_periodic_gpu(&dpB);

        float lambda = d_fepparams->lambda_q;

        float ddphi = -((1 - lambda) * kA * dpA + lambda * kB * dpB);

        do_dih_fup_gpu<calcVir>(ai, aj, ak, al, -ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc,
                                pbcAiuc, gm_xq, t1, t2, t3);

        if (calcEner)
        {
            *vtot_loc += -0.5f * ddphi * dpA;
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void pairs_gpu(const int       i,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams iparams[],
                          const float4    gm_xq[],
                          fvec            gm_f[],
                          fvec            sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc,
                          const float     scale_factor,
                          float*          vtotVdw_loc,
                          float*          vtotElec_loc)
{
    if (i < numBonds)
    {
        int3 pairData = *(int3*)(d_forceatoms + 3 * i);
        int  type     = pairData.x;
        int  ai       = pairData.y;
        int  aj       = pairData.z;

        float qq  = gm_xq[ai].w * gm_xq[aj].w;
        float c6  = iparams[type].lj14.c6A;
        float c12 = iparams[type].lj14.c12A;

        /* Do we need to apply full periodic boundary conditions? */
        fvec dr;
        int  fshift_index = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[aj], dr);

        float r2    = norm2_gpu(dr);
        float rinv  = rsqrtf(r2);
        float rinv2 = rinv * rinv;
        float rinv6 = rinv2 * rinv2 * rinv2;

        /* Calculate the Coulomb force * r */
        float velec = scale_factor * qq * rinv;

        /* Calculate the LJ force * r and add it to the Coulomb part */
        float fr = (12.0f * c12 * rinv6 - 6.0f * c6) * rinv6 + velec;

        float finvr = fr * rinv2;
        fvec  f;
        svmul_gpu(finvr, dr, f);

        /* Add the forces */
#pragma unroll
        for (int m = 0; m < DIM; m++)
        {
            atomicAdd(&gm_f[ai][m], f[m]);
            atomicAdd(&gm_f[aj][m], -f[m]);
            if (calcVir && fshift_index != CENTRAL)
            {
                atomicAdd(&sm_fShiftLoc[fshift_index][m], f[m]);
                atomicAdd(&sm_fShiftLoc[CENTRAL][m], -f[m]);
            }
        }

        if (calcEner)
        {
            *vtotVdw_loc += (c12 * rinv6 - c6) * rinv6;
            *vtotElec_loc += velec;
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void pairs_fep_gpu(const int       i,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams iparams[],
                          gmx::BondedFepParameters* d_fepparams,
                          const float4    gm_xq[],
                          const float     gm_qA[],
                          const float     gm_qB[],
                          fvec            gm_f[],
                          fvec            sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc,
                          const float     scale_factor,
                          float*          vtotVdw_loc,
                          float*          vtotElec_loc)
{
    const bool bFEP = d_fepparams->bFEP;
    bool       bFEPpair = 0;
    const float alpha_coul = d_fepparams->alpha_coul;
    const float alpha_vdw  = d_fepparams->alpha_vdw;
    float alpha_coul_eff   = alpha_coul;
    float alpha_vdw_eff    = alpha_vdw;
    const bool useSoftCore = (alpha_vdw != 0.0);
    const bool useScBetaNO = (alpha_coul == 0.0);
    const float sigma6_def = d_fepparams->sc_sigma6;
    const float sigma6_min = d_fepparams->sc_sigma6_min;
    const float lambda_q   = d_fepparams->lambda_q;
    const float _lambda_q  = 1 - lambda_q;
    const float lambda_v   = d_fepparams->lambda_v;
    const float _lambda_v  = 1 - lambda_v;

    const float lfac_coul[2] = {lambda_q, _lambda_q};
    const float lfac_vdw[2]  = {lambda_v, _lambda_v};
    const float LFC[2]       = {_lambda_q, lambda_q};
    const float LFV[2]       = {_lambda_v, lambda_v};

    float FscalC[2], FscalV[2], Vcoul[2], Vvdw[2];
    float rinvC, rinvV, r2C, r2V, rpinvC, rpinvV;

    if (i < numBonds)
    {
        int3 pairData = *(int3*)(d_forceatoms + 3 * i);
        int  type     = pairData.x;
        int  ai       = pairData.y;
        int  aj       = pairData.z;

        float qq[2]  = {gm_qA[ai] * gm_qA[aj], gm_qB[ai] * gm_qB[aj]};
        float c6AB[2]  = {iparams[type].lj14.c6A, iparams[type].lj14.c6B};
        float c12AB[2] = {iparams[type].lj14.c12A, iparams[type].lj14.c12B};
        float sigma6[2];
        float velec = 0;
        float vlj   = 0;
        float finvr = 0;

        if (qq[0] == qq[1] && c6AB[0] == c6AB[1] && c12AB[0] == c12AB[1]) bFEPpair = 1;
        else bFEPpair = 0;

        /* Do we need to apply full periodic boundary conditions? */
        fvec dr;
        int  fshift_index = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[aj], dr);

        float r2    = norm2_gpu(dr);
        float rpm2  = r2 * r2;
        float rp    = rpm2 * r2;
        float rinv  = rsqrtf(r2);
        float rinv2 = rinv * rinv;
        float rinv6 = rinv2 * rinv2 * rinv2;

        if (bFEPpair)
        {
            for (int k = 0; k < 2; k++)
            {
                FscalC[k] = 0;
                FscalV[k] = 0;
                if (calcEner)
                {
                    Vcoul[k]  = 0;
                    Vvdw[k]   = 0;
                }
                if ((qq[k] != 0) || (c6AB[k] != 0) || (c12AB[k] != 0))
                {
                    if ((c12AB[0] == 0 || c12AB[1] == 0) && (useSoftCore))
                    {
                        if (c6AB[k] == 0) sigma6[k] = 0;
                        else sigma6[k] = c12AB[k] / c6AB[k];
                        alpha_vdw_eff  = alpha_vdw;
                        alpha_coul_eff = (useScBetaNO ? alpha_vdw_eff : alpha_coul);
                        if (sigma6[k] == 0)          sigma6[k] = sigma6_def;
                        if (sigma6[k] < sigma6_min)  sigma6[k] = sigma6_min;

                        if (useScBetaNO)
                        {
                            rpinvC = 1.0f / (alpha_coul_eff * lfac_coul[k] * sigma6[k] + rp);
                        }
                        // else if (softCoreTreatment == SoftCoreTreatment::RPower6_2)
                        // {
                        //     rpinvC = one / (alpha_coul_eff * lfac_coul[i] + rpc);
                        //     sqRoot<softCoreTreatment>(rpinvC, &rinvC, &rC);
                        // }
                        else
                        {
                            rpinvC = 1.0f / (alpha_coul_eff * lfac_coul[k] * sigma6[k] + rp);
                        }
                        r2C    = rcbrt(rpinvC);
                        rinvC  = rsqrt(r2C);
                
                        if ((alpha_coul_eff != alpha_vdw_eff) || (!useScBetaNO))
                        {
                            rpinvV = 1.0f / (alpha_vdw_eff * lfac_vdw[k] * sigma6[k] + rp);
                            r2V    = rcbrt(rpinvV);
                            rinvV  = rsqrt(r2V);
                        }
                        else
                        {
                            /* We can avoid one expensive pow and one / operation */
                            rpinvV = rpinvC;
                            rinvV  = rinvC;
                        }
                    }
                    else
                    {
                        rpinvC = rinv6;
                        rinvC  = rinv;
                        rpinvV = rinv6;
                        rinvV  = rinv;
                    }
// #        endif /* EXCLUSION_FORCES */
                    // #    if !defined LJ_COMB_LB || defined CALC_ENERGIES
                                    
                    if (c6AB[k] != 0 || c12AB[k] != 0)
                    {
                        float Vvdw6 = c6AB[k] * rpinvV;
                        float Vvdw12= c12AB[k] * rpinvV * rpinvV;
                        FscalV[k] = 12.0f * Vvdw12 - 6.0f * Vvdw6;
                        if (calcEner)
                        {
                            Vvdw[k] = Vvdw12 - Vvdw6;
                        }
                    }

                    if (qq[k] != 0)
                    {
                        FscalC[k] = scale_factor * qq[k] * rinvC;
                        Vcoul[k] = FscalC[k];
                    }
                            // if ((c6AB[0] != 0 || c6AB[1] != 0 || c12AB[0]!=0 || c12AB[1]!=0 || qq[0]!=0 || qq[1]!=0) && (k == 1))
                            // printf("interaction [%d-%d], r2=[%e], mask=%f, FscalC=[%e, %e], FscalV=[%e, %e]\n", ai, aj, r2, int_bit, FscalC[0] * rpm2, FscalC[1] * rpm2, FscalV[0] * rpm2, FscalV[1] * rpm2);
                    FscalC[k] *= rpinvC;
                    FscalV[k] *= rpinvV;
                }
            }
            for (int k = 0; k < 2; k++)
            {
                if (calcEner)
                {
                    velec += LFC[k] * Vcoul[k];
                    vlj += LFV[k] * Vvdw[k];
                }
                finvr += LFC[k] * FscalC[k] * rpm2;
                finvr += LFV[k] * FscalV[k] * rpm2;
            }
        }
        else
        {
            /* Calculate the Coulomb force * r */
            velec = scale_factor * qq[0] * rinv;
            vlj   = (c12AB[0] * rinv6 - c6AB[0]) * rinv6;

            /* Calculate the LJ force * r and add it to the Coulomb part */
            float fr = (12.0f * c12AB[0] * rinv6 - 6.0f * c6AB[0]) * rinv6 + velec;
            finvr = fr * rinv2;
        }

        fvec  f;
        svmul_gpu(finvr, dr, f);

        /* Add the forces */
#pragma unroll
        for (int m = 0; m < DIM; m++)
        {
            atomicAdd(&gm_f[ai][m], f[m]);
            atomicAdd(&gm_f[aj][m], -f[m]);
            if (calcVir && fshift_index != CENTRAL)
            {
                atomicAdd(&sm_fShiftLoc[fshift_index][m], f[m]);
                atomicAdd(&sm_fShiftLoc[CENTRAL][m], -f[m]);
            }
        }

        if (calcEner)
        {
            *vtotVdw_loc += vlj;
            *vtotElec_loc += velec;
        }
    }
}

namespace gmx
{

template<bool calcVir, bool calcEner>
__global__ void exec_kernel_gpu(BondedCudaKernelParameters kernelParams)
{
    assert(blockDim.y == 1 && blockDim.z == 1);
    const int  tid          = blockIdx.x * blockDim.x + threadIdx.x;
    float      vtot_loc     = 0;
    float      vtotVdw_loc  = 0;
    float      vtotElec_loc = 0;
    __shared__ fvec sm_fShiftLoc[SHIFTS];

    if (calcVir)
    {
        if (threadIdx.x < SHIFTS)
        {
            sm_fShiftLoc[threadIdx.x][XX] = 0.0f;
            sm_fShiftLoc[threadIdx.x][YY] = 0.0f;
            sm_fShiftLoc[threadIdx.x][ZZ] = 0.0f;
        }
        __syncthreads();
    }

    int  fType;
    bool threadComputedPotential = false;
#pragma unroll
    for (int j = 0; j < numFTypesOnGpu; j++)
    {
        if (tid >= kernelParams.fTypeRangeStart[j] && tid <= kernelParams.fTypeRangeEnd[j])
        {
            const int      numBonds = kernelParams.numFTypeBonds[j];
            int            fTypeTid = tid - kernelParams.fTypeRangeStart[j];
            const t_iatom* iatoms   = kernelParams.d_iatoms[j];
            fType                   = kernelParams.fTypesOnGpu[j];
            if (calcEner)
            {
                threadComputedPotential = true;
            }

            switch (fType)
            {
                case F_BONDS:
                    // bonds_gpu<calcVir, calcEner>(fTypeTid, &vtot_loc, numBonds, iatoms,
                    //                              kernelParams.d_forceParams, kernelParams.d_xq,
                    //                              kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    bonds_fep_gpu<calcVir, calcEner>(fTypeTid, &vtot_loc, numBonds, iatoms,
                                                 kernelParams.d_forceParams, kernelParams.d_fepParams, kernelParams.d_xq,
                                                 kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_ANGLES:
                    // angles_gpu<calcVir, calcEner>(
                    //         fTypeTid, &vtot_loc, numBonds, iatoms, kernelParams.d_forceParams,
                    //         kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    angles_fep_gpu<calcVir, calcEner>(
                            fTypeTid, &vtot_loc, numBonds, iatoms, kernelParams.d_forceParams, kernelParams.d_fepParams,
                            kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_UREY_BRADLEY:
                    // urey_bradley_gpu<calcVir, calcEner>(
                    //         fTypeTid, &vtot_loc, numBonds, iatoms, kernelParams.d_forceParams,
                    //         kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    urey_bradley_fep_gpu<calcVir, calcEner>(
                            fTypeTid, &vtot_loc, numBonds, iatoms, kernelParams.d_forceParams, kernelParams.d_fepParams,
                            kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_PDIHS:
                case F_PIDIHS:
                    // pdihs_gpu<calcVir, calcEner>(fTypeTid, &vtot_loc, numBonds, iatoms,
                    //                              kernelParams.d_forceParams, kernelParams.d_xq,
                    //                              kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    pdihs_fep_gpu<calcVir, calcEner>(fTypeTid, &vtot_loc, numBonds, iatoms,
                                                 kernelParams.d_forceParams, kernelParams.d_fepParams, kernelParams.d_xq,
                                                 kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_RBDIHS:
                    // rbdihs_gpu<calcVir, calcEner>(
                    //         fTypeTid, &vtot_loc, numBonds, iatoms, kernelParams.d_forceParams,
                    //         kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    rbdihs_fep_gpu<calcVir, calcEner>(
                            fTypeTid, &vtot_loc, numBonds, iatoms, kernelParams.d_forceParams, kernelParams.d_fepParams,
                            kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_IDIHS:
                    // idihs_gpu<calcVir, calcEner>(fTypeTid, &vtot_loc, numBonds, iatoms,
                    //                              kernelParams.d_forceParams, kernelParams.d_xq,
                    //                              kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    idihs_fep_gpu<calcVir, calcEner>(fTypeTid, &vtot_loc, numBonds, iatoms,
                                                 kernelParams.d_forceParams, kernelParams.d_fepParams, kernelParams.d_xq,
                                                 kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_LJ14:
                    // pairs_gpu<calcVir, calcEner>(fTypeTid, numBonds, iatoms, kernelParams.d_forceParams,
                    //                              kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc,
                    //                              kernelParams.pbcAiuc, kernelParams.scaleFactor,
                    //                              &vtotVdw_loc, &vtotElec_loc);
                    pairs_fep_gpu<calcVir, calcEner>(fTypeTid, numBonds, iatoms, kernelParams.d_forceParams, kernelParams.d_fepParams,
                                                 kernelParams.d_xq, kernelParams.d_qA, kernelParams.d_qB, kernelParams.d_f, sm_fShiftLoc,
                                                 kernelParams.pbcAiuc, kernelParams.scaleFactor,
                                                 &vtotVdw_loc, &vtotElec_loc);
                    break;
            }
            break;
        }
    }

    if (threadComputedPotential)
    {
        float* vtotVdw  = kernelParams.d_vTot + F_LJ14;
        float* vtotElec = kernelParams.d_vTot + F_COUL14;
        atomicAdd(kernelParams.d_vTot + fType, vtot_loc);
        atomicAdd(vtotVdw, vtotVdw_loc);
        atomicAdd(vtotElec, vtotElec_loc);
    }
    /* Accumulate shift vectors from shared memory to global memory on the first SHIFTS threads of the block. */
    if (calcVir)
    {
        __syncthreads();
        if (threadIdx.x < SHIFTS)
        {
            fvec_inc_atomic(kernelParams.d_fShift[threadIdx.x], sm_fShiftLoc[threadIdx.x]);
        }
    }
}


/*-------------------------------- End CUDA kernels-----------------------------*/


template<bool calcVir, bool calcEner>
void GpuBonded::Impl::launchKernel(const t_forcerec* fr, const matrix box)
{
    GMX_ASSERT(haveInteractions_,
               "Cannot launch bonded GPU kernels unless bonded GPU work was scheduled");
    static_assert(TPB_BONDED >= SHIFTS,
                  "TPB_BONDED must be >= SHIFTS for the virial kernel (calcVir=true)");

    PbcAiuc pbcAiuc;
    setPbcAiuc(fr->bMolPBC ? ePBC2npbcdim(fr->ePBC) : 0, box, &pbcAiuc);

    int fTypeRangeEnd = kernelParams_.fTypeRangeEnd[numFTypesOnGpu - 1];

    if (fTypeRangeEnd < 0)
    {
        return;
    }

    KernelLaunchConfig config;
    config.blockSize[0] = TPB_BONDED;
    config.blockSize[1] = 1;
    config.blockSize[2] = 1;
    config.gridSize[0]  = (fTypeRangeEnd + TPB_BONDED) / TPB_BONDED;
    config.gridSize[1]  = 1;
    config.gridSize[2]  = 1;
    config.sharedMemorySize = SHIFTS * sizeof(float3);
    config.stream       = stream_;

    auto kernelPtr            = exec_kernel_gpu<calcVir, calcEner>;
    kernelParams_.scaleFactor = fr->ic->epsfac * fr->fudgeQQ;
    kernelParams_.pbcAiuc     = pbcAiuc;

    const auto kernelArgs = prepareGpuKernelArguments(kernelPtr, config, &kernelParams_);

    launchGpuKernel(kernelPtr, config, nullptr, "exec_kernel_gpu<calcVir, calcEner>", kernelArgs);
}

void GpuBonded::launchKernel(const t_forcerec* fr, const gmx::StepWorkload& stepWork, const matrix box)
{
    if (stepWork.computeEnergy)
    {
        // When we need the energy, we also need the virial
        impl_->launchKernel<true, true>(fr, box);
    }
    else if (stepWork.computeVirial)
    {
        impl_->launchKernel<true, false>(fr, box);
    }
    else
    {
        impl_->launchKernel<false, false>(fr, box);
    }
}

} // namespace gmx
