#include "hip/hip_runtime.h"
/*
 * This file is part of the GROMACS molecular simulation package.
 *
 * Copyright (c) 2018,2019,2020,2021, by the GROMACS development team, led by
 * Mark Abraham, David van der Spoel, Berk Hess, and Erik Lindahl,
 * and including many others, as listed in the AUTHORS file in the
 * top-level source directory and at http://www.gromacs.org.
 *
 * GROMACS is free software; you can redistribute it and/or
 * modify it under the terms of the GNU Lesser General Public License
 * as published by the Free Software Foundation; either version 2.1
 * of the License, or (at your option) any later version.
 *
 * GROMACS is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the GNU
 * Lesser General Public License for more details.
 *
 * You should have received a copy of the GNU Lesser General Public
 * License along with GROMACS; if not, see
 * http://www.gnu.org/licenses, or write to the Free Software Foundation,
 * Inc., 51 Franklin Street, Fifth Floor, Boston, MA  02110-1301  USA.
 *
 * If you want to redistribute modifications to GROMACS, please
 * consider that scientific software is very special. Version
 * control is crucial - bugs must be traceable. We will be happy to
 * consider code for inclusion in the official distribution, but
 * derived work must not be called official GROMACS. Details are found
 * in the README & COPYING files - if they are missing, get the
 * official version at http://www.gromacs.org.
 *
 * To help us fund GROMACS development, we humbly ask that you cite
 * the research papers on the package. Check out http://www.gromacs.org.
 */
/*! \internal \file
 *
 * \brief Implements CUDA bonded functionality
 *
 * \author Jon Vincent <jvincent@nvidia.com>
 * \author Magnus Lundborg <lundborg.magnus@gmail.com>
 * \author Berk Hess <hess@kth.se>
 * \author Szilárd Páll <pall.szilard@gmail.com>
 * \author Alan Gray <alang@nvidia.com>
 * \author Mark Abraham <mark.j.abraham@gmail.com>
 *
 * \ingroup module_listed_forces
 */

#include "gmxpre.h"

#include <cassert>

#include <hip/hip_math_constants.h>

#include "gromacs/gpu_utils/cudautils.cuh"
#include "gromacs/gpu_utils/typecasts.cuh"
#include "gromacs/gpu_utils/vectype_ops.cuh"
#include "gromacs/listed_forces/gpubonded.h"
#include "gromacs/math/units.h"
#include "gromacs/mdlib/force_flags.h"
#include "gromacs/mdtypes/interaction_const.h"
#include "gromacs/mdtypes/simulation_workload.h"
#include "gromacs/pbcutil/pbc_aiuc_cuda.cuh"
#include "gromacs/timing/wallcycle.h"
#include "gromacs/utility/gmxassert.h"

#include "gpubonded_impl.h"

struct BondedFepParameters;

#if defined(_MSVC)
#    include <limits>
#endif

/*-------------------------------- CUDA kernels-------------------------------- */
/*------------------------------------------------------------------------------*/

#define CUDA_DEG2RAD_F (HIP_PI_F / 180.0f)

/*---------------- BONDED CUDA kernels--------------*/

/* Harmonic */
__device__ __forceinline__ static void
           harmonic_gpu(const float kA, const float xA, const float x, float* V, float* F)
{
    constexpr float half = 0.5f;
    float           dx, dx2;

    dx  = x - xA;
    dx2 = dx * dx;

    *F = -kA * dx;
    *V = half * kA * dx2;
}

__device__ __forceinline__ static void
           harmonic_fep_gpu(const float kA, const float kB, const float xA, const float xB, const float x, const float lambda, float* V, float* F)
{
    constexpr float half = 0.5f;
    float           L1, kk, x0, dx, dx2;
    float           v, f, dvdlambda;

    L1 = 1.0 - lambda;
    kk = L1 * kA + lambda * kB;
    x0 = L1 * xA + lambda * xB;

    dx  = x - x0;
    dx2 = dx * dx;

    f         = -kk * dx;
    v         = half * kk * dx2;
    dvdlambda = half * (kB - kA) * dx2 + (xA - xB) * kk * dx;

    *F = f;
    *V = v;
}

template<bool calcVir, bool calcEner>
__device__ void bonds_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          const float4    gm_xq[],
                          float3          gm_f[],
                          float3          sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int3 bondData = *(int3*)(d_forceatoms + 3 * i);
        int  type     = bondData.x;
        int  ai       = bondData.y;
        int  aj       = bondData.z;

        /* dx = xi - xj, corrected for periodic boundary conditions. */
        float3 dx;
        int    ki = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[aj], dx);

        float dr2 = norm2(dx);
        float dr  = sqrt(dr2);

        float vbond;
        float fbond;
        harmonic_gpu(d_forceparams[type].harmonic.krA, d_forceparams[type].harmonic.rA, dr, &vbond, &fbond);

        if (calcEner)
        {
            *vtot_loc += vbond;
        }

        if (dr2 != 0.0f)
        {
            fbond *= rsqrtf(dr2);

            float3 fij = fbond * dx;
            atomicAdd(&gm_f[ai], fij);
            atomicAdd(&gm_f[aj], -fij);
            if (calcVir && ki != CENTRAL)
            {
                atomicAdd(&sm_fShiftLoc[ki], fij);
                atomicAdd(&sm_fShiftLoc[CENTRAL], -fij);
            }
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void bonds_fep_gpu(const int                 i,
                              float*                    vtot_loc,
                              const int                 numBonds,
                              const t_iatom             d_forceatoms[],
                              const t_iparams           d_forceparams[],
                              gmx::BondedFepParameters* d_fepparams,
                              const float4              gm_xq[],
                              float3                      gm_f[],
                              float3                      sm_fShiftLoc[],
                              const PbcAiuc             pbcAiuc)
{
    if (i < numBonds)
    {
        int3 bondData = *(int3*)(d_forceatoms + 3 * i);
        int  type     = bondData.x;
        int  ai       = bondData.y;
        int  aj       = bondData.z;

        /* dx = xi - xj, corrected for periodic boundary conditions. */
        float3 dx;
        int  ki = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[aj], dx);

        float dr2 = norm2(dx);
        float dr  = sqrt(dr2);

        float vbond;
        float fbond;
        harmonic_fep_gpu(d_forceparams[type].harmonic.krA, d_forceparams[type].harmonic.krB,
                         d_forceparams[type].harmonic.rA, d_forceparams[type].harmonic.rB, dr,
                         d_fepparams->lambda_q, &vbond, &fbond);

        if (calcEner)
        {
            *vtot_loc += vbond;
        }

        if (dr2 != 0.0f)
        {
            fbond *= rsqrtf(dr2);

            float3 fij = fbond * dx;
            atomicAdd(&gm_f[ai], fij);
            atomicAdd(&gm_f[aj], -fij);
            if (calcVir && ki != CENTRAL)
            {
                atomicAdd(&sm_fShiftLoc[ki], fij);
                atomicAdd(&sm_fShiftLoc[CENTRAL], -fij);
            }
        }
    }
}

template<bool returnShift>
__device__ __forceinline__ static float bond_angle_gpu(const float4   xi,
                                                       const float4   xj,
                                                       const float4   xk,
                                                       const PbcAiuc& pbcAiuc,
                                                       float3*        r_ij,
                                                       float3*        r_kj,
                                                       float*         costh,
                                                       int*           t1,
                                                       int*           t2)
/* Return value is the angle between the bonds i-j and j-k */
{
    *t1 = pbcDxAiuc<returnShift>(pbcAiuc, xi, xj, *r_ij);
    *t2 = pbcDxAiuc<returnShift>(pbcAiuc, xk, xj, *r_kj);

    *costh   = cos_angle(*r_ij, *r_kj);
    float th = acosf(*costh);

    return th;
}

template<bool calcVir, bool calcEner>
__device__ void angles_gpu(const int       i,
                           float*          vtot_loc,
                           const int       numBonds,
                           const t_iatom   d_forceatoms[],
                           const t_iparams d_forceparams[],
                           const float4    gm_xq[],
                           float3          gm_f[],
                           float3          sm_fShiftLoc[],
                           const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int4 angleData = *(int4*)(d_forceatoms + 4 * i);
        int  type      = angleData.x;
        int  ai        = angleData.y;
        int  aj        = angleData.z;
        int  ak        = angleData.w;

        float3 r_ij;
        float3 r_kj;
        float  cos_theta;
        int    t1;
        int    t2;
        float  theta = bond_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], pbcAiuc, &r_ij,
                                              &r_kj, &cos_theta, &t1, &t2);

        float va;
        float dVdt;
        harmonic_gpu(d_forceparams[type].harmonic.krA,
                     d_forceparams[type].harmonic.rA * CUDA_DEG2RAD_F, theta, &va, &dVdt);

        if (calcEner)
        {
            *vtot_loc += va;
        }

        float cos_theta2 = cos_theta * cos_theta;
        if (cos_theta2 < 1.0f)
        {
            float st    = dVdt * rsqrtf(1.0f - cos_theta2);
            float sth   = st * cos_theta;
            float nrij2 = norm2(r_ij);
            float nrkj2 = norm2(r_kj);

            float nrij_1 = rsqrtf(nrij2);
            float nrkj_1 = rsqrtf(nrkj2);

            float cik = st * nrij_1 * nrkj_1;
            float cii = sth * nrij_1 * nrij_1;
            float ckk = sth * nrkj_1 * nrkj_1;

            float3 f_i = cii * r_ij - cik * r_kj;
            float3 f_k = ckk * r_kj - cik * r_ij;
            float3 f_j = -f_i - f_k;

            atomicAdd(&gm_f[ai], f_i);
            atomicAdd(&gm_f[aj], f_j);
            atomicAdd(&gm_f[ak], f_k);

            if (calcVir)
            {
                atomicAdd(&sm_fShiftLoc[t1], f_i);
                atomicAdd(&sm_fShiftLoc[CENTRAL], f_j);
                atomicAdd(&sm_fShiftLoc[t2], f_k);
            }
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void angles_fep_gpu(const int                 i,
                               float*                    vtot_loc,
                               const int                 numBonds,
                               const t_iatom             d_forceatoms[],
                               const t_iparams           d_forceparams[],
                               gmx::BondedFepParameters* d_fepparams,
                               const float4              gm_xq[],
                               float3                      gm_f[],
                               float3                      sm_fShiftLoc[],
                               const PbcAiuc             pbcAiuc)
{
    if (i < numBonds)
    {
        int4 angleData = *(int4*)(d_forceatoms + 4 * i);
        int  type      = angleData.x;
        int  ai        = angleData.y;
        int  aj        = angleData.z;
        int  ak        = angleData.w;

        float3  r_ij;
        float3  r_kj;
        float cos_theta;
        int   t1;
        int   t2;
        float theta = bond_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], pbcAiuc, &r_ij, &r_kj,
                                              &cos_theta, &t1, &t2);

        float va;
        float dVdt;
        harmonic_fep_gpu(d_forceparams[type].harmonic.krA, d_forceparams[type].harmonic.krB,
                         d_forceparams[type].harmonic.rA * CUDA_DEG2RAD_F,
                         d_forceparams[type].harmonic.rB * CUDA_DEG2RAD_F, theta,
                         d_fepparams->lambda_q, &va, &dVdt);

        if (calcEner)
        {
            *vtot_loc += va;
        }

        float cos_theta2 = cos_theta * cos_theta;
        if (cos_theta2 < 1.0f)
        {
            float st    = dVdt * rsqrtf(1.0f - cos_theta2);
            float sth   = st * cos_theta;
            float nrij2 = norm2(r_ij);
            float nrkj2 = norm2(r_kj);

            float nrij_1 = rsqrtf(nrij2);
            float nrkj_1 = rsqrtf(nrkj2);

            float cik = st * nrij_1 * nrkj_1;
            float cii = sth * nrij_1 * nrij_1;
            float ckk = sth * nrkj_1 * nrkj_1;

            float3 f_i;
            float3 f_k;
            float3 f_j;

            f_i = cii * r_ij - cik * r_kj;
            f_k = ckk * r_kj - cik * r_ij;
            f_j = -f_i - f_k;
            atomicAdd(&gm_f[ai], f_i);
            atomicAdd(&gm_f[aj], f_j);
            atomicAdd(&gm_f[ak], f_k);
            if (calcVir)
            {
                atomicAdd(&sm_fShiftLoc[t1], f_i);
                atomicAdd(&sm_fShiftLoc[CENTRAL], f_j);
                atomicAdd(&sm_fShiftLoc[t2], f_k);
            }
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void urey_bradley_gpu(const int       i,
                                 float*          vtot_loc,
                                 const int       numBonds,
                                 const t_iatom   d_forceatoms[],
                                 const t_iparams d_forceparams[],
                                 const float4    gm_xq[],
                                 float3          gm_f[],
                                 float3          sm_fShiftLoc[],
                                 const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int4 ubData = *(int4*)(d_forceatoms + 4 * i);
        int  type   = ubData.x;
        int  ai     = ubData.y;
        int  aj     = ubData.z;
        int  ak     = ubData.w;

        float th0A = d_forceparams[type].u_b.thetaA * CUDA_DEG2RAD_F;
        float kthA = d_forceparams[type].u_b.kthetaA;
        float r13A = d_forceparams[type].u_b.r13A;
        float kUBA = d_forceparams[type].u_b.kUBA;

        float3 r_ij;
        float3 r_kj;
        float  cos_theta;
        int    t1;
        int    t2;
        float  theta = bond_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], pbcAiuc, &r_ij,
                                              &r_kj, &cos_theta, &t1, &t2);

        float va;
        float dVdt;
        harmonic_gpu(kthA, th0A, theta, &va, &dVdt);

        if (calcEner)
        {
            *vtot_loc += va;
        }

        float3 r_ik;
        int    ki = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[ak], r_ik);

        float dr2 = norm2(r_ik);
        float dr  = dr2 * rsqrtf(dr2);

        float vbond;
        float fbond;
        harmonic_gpu(kUBA, r13A, dr, &vbond, &fbond);

        float cos_theta2 = cos_theta * cos_theta;
        if (cos_theta2 < 1.0f)
        {
            float st  = dVdt * rsqrtf(1.0f - cos_theta2);
            float sth = st * cos_theta;

            float nrkj2 = norm2(r_kj);
            float nrij2 = norm2(r_ij);

            float cik = st * rsqrtf(nrkj2 * nrij2);
            float cii = sth / nrij2;
            float ckk = sth / nrkj2;

            float3 f_i = cii * r_ij - cik * r_kj;
            float3 f_k = ckk * r_kj - cik * r_ij;
            float3 f_j = -f_i - f_k;

            atomicAdd(&gm_f[ai], f_i);
            atomicAdd(&gm_f[aj], f_j);
            atomicAdd(&gm_f[ak], f_k);

            if (calcVir)
            {
                atomicAdd(&sm_fShiftLoc[t1], f_i);
                atomicAdd(&sm_fShiftLoc[CENTRAL], f_j);
                atomicAdd(&sm_fShiftLoc[t2], f_k);
            }
        }

        /* Time for the bond calculations */
        if (dr2 != 0.0f)
        {
            if (calcEner)
            {
                *vtot_loc += vbond;
            }

            fbond *= rsqrtf(dr2);

            float3 fik = fbond * r_ik;
            atomicAdd(&gm_f[ai], fik);
            atomicAdd(&gm_f[ak], -fik);

            if (calcVir && ki != CENTRAL)
            {
                atomicAdd(&sm_fShiftLoc[ki], fik);
                atomicAdd(&sm_fShiftLoc[CENTRAL], -fik);
            }
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void urey_bradley_fep_gpu(const int                 i,
                                     float*                    vtot_loc,
                                     const int                 numBonds,
                                     const t_iatom             d_forceatoms[],
                                     const t_iparams           d_forceparams[],
                                     gmx::BondedFepParameters* d_fepparams,
                                     const float4              gm_xq[],
                                     float3                      gm_f[],
                                     float3                      sm_fShiftLoc[],
                                     const PbcAiuc             pbcAiuc)
{
    if (i < numBonds)
    {
        int4 ubData = *(int4*)(d_forceatoms + 4 * i);
        int  type   = ubData.x;
        int  ai     = ubData.y;
        int  aj     = ubData.z;
        int  ak     = ubData.w;

        float th0A = d_forceparams[type].u_b.thetaA * CUDA_DEG2RAD_F;
        float kthA = d_forceparams[type].u_b.kthetaA;
        float r13A = d_forceparams[type].u_b.r13A;
        float kUBA = d_forceparams[type].u_b.kUBA;

        float th0B = d_forceparams[type].u_b.thetaB * CUDA_DEG2RAD_F;
        float kthB = d_forceparams[type].u_b.kthetaB;
        float r13B = d_forceparams[type].u_b.r13B;
        float kUBB = d_forceparams[type].u_b.kUBB;

        float3  r_ij;
        float3  r_kj;
        float cos_theta;
        int   t1;
        int   t2;
        float theta = bond_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], pbcAiuc, &r_ij, &r_kj,
                                              &cos_theta, &t1, &t2);

        float va;
        float dVdt;
        // harmonic_gpu(kthA, th0A, theta, &va, &dVdt);
        harmonic_fep_gpu(kthA, kthB, th0A, th0B, theta, d_fepparams->lambda_q, &va, &dVdt);

        if (calcEner)
        {
            *vtot_loc += va;
        }

        float3 r_ik;
        int  ki = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[ak], r_ik);

        float dr2 = norm2(r_ik);
        float dr  = dr2 * rsqrtf(dr2);

        float vbond;
        float fbond;
        // harmonic_gpu(kUBA, r13A, dr, &vbond, &fbond);
        harmonic_fep_gpu(kUBA, kUBB, r13A, r13B, dr, d_fepparams->lambda_q, &vbond, &fbond);

        float cos_theta2 = cos_theta * cos_theta;
        if (cos_theta2 < 1.0f)
        {
            float st  = dVdt * rsqrtf(1.0f - cos_theta2);
            float sth = st * cos_theta;

            float nrkj2 = norm2(r_kj);
            float nrij2 = norm2(r_ij);

            float cik = st * rsqrtf(nrkj2 * nrij2);
            float cii = sth / nrij2;
            float ckk = sth / nrkj2;

            float3 f_i;
            float3 f_j;
            float3 f_k;
            f_i = cii * r_ij - cik * r_kj;
            f_k = ckk * r_kj - cik * r_ij;
            f_j = -f_i - f_k;
            atomicAdd(&gm_f[ai], f_i);
            atomicAdd(&gm_f[aj], f_j);
            atomicAdd(&gm_f[ak], f_k);
            if (calcVir)
            {
                atomicAdd(&sm_fShiftLoc[t1], f_i);
                atomicAdd(&sm_fShiftLoc[CENTRAL], f_j);
                atomicAdd(&sm_fShiftLoc[t2], f_k);
            }
        }

        /* Time for the bond calculations */
        if (dr2 != 0.0f)
        {
            if (calcEner)
            {
                *vtot_loc += vbond;
            }

            fbond *= rsqrtf(dr2);

            float3 fik = fbond * r_ik;
            atomicAdd(&gm_f[ai], fik);
            atomicAdd(&gm_f[ak], -fik);

            if (calcVir && ki != CENTRAL)
            {
                atomicAdd(&sm_fShiftLoc[ki], fik);
                atomicAdd(&sm_fShiftLoc[CENTRAL], -fik);
            }
        }
    }
}

template<bool returnShift, typename T>
__device__ __forceinline__ static float dih_angle_gpu(const T        xi,
                                                      const T        xj,
                                                      const T        xk,
                                                      const T        xl,
                                                      const PbcAiuc& pbcAiuc,
                                                      float3*        r_ij,
                                                      float3*        r_kj,
                                                      float3*        r_kl,
                                                      float3*        m,
                                                      float3*        n,
                                                      int*           t1,
                                                      int*           t2,
                                                      int*           t3)
{
    *t1 = pbcDxAiuc<returnShift>(pbcAiuc, xi, xj, *r_ij);
    *t2 = pbcDxAiuc<returnShift>(pbcAiuc, xk, xj, *r_kj);
    *t3 = pbcDxAiuc<returnShift>(pbcAiuc, xk, xl, *r_kl);

    *m         = cprod(*r_ij, *r_kj);
    *n         = cprod(*r_kj, *r_kl);
    float phi  = gmx_angle(*m, *n);
    float ipr  = iprod(*r_ij, *n);
    float sign = (ipr < 0.0f) ? -1.0f : 1.0f;
    phi        = sign * phi;

    return phi;
}


__device__ __forceinline__ static void
           dopdihs_gpu(const float cpA, const float phiA, const int mult, const float phi, float* v, float* f)
{
    float mdphi, sdphi;

    mdphi = mult * phi - phiA * CUDA_DEG2RAD_F;
    sdphi = sinf(mdphi);
    *v    = cpA * (1.0f + cosf(mdphi));
    *f    = -cpA * mult * sdphi;
}


__device__ __forceinline__ static void
           dopdihs_fep_gpu(const float cpA, const float cpB, const float phiA, const float phiB, const int mult, const float phi, const float lambda, float* v, float* f)
{
    float mdphiA, sdphiA, mdphiB, sdphiB;
    mdphiA = mult * phi - phiA * CUDA_DEG2RAD_F;
    mdphiB = mult * phi - phiB * CUDA_DEG2RAD_F;
    sdphiA = sinf(mdphiA);
    sdphiB = sinf(mdphiB);
    *v    = (1 - lambda) * cpA * (1.0f + cosf(mdphiA)) + lambda * cpB * (1.0f + cosf(mdphiB));
    *f    = -((1 - lambda) * cpA * mult * sdphiA + lambda * cpB * mult * sdphiB);
}

template<bool calcVir>
__device__ static void do_dih_fup_gpu(const int      i,
                                      const int      j,
                                      const int      k,
                                      const int      l,
                                      const float    ddphi,
                                      const float3   r_ij,
                                      const float3   r_kj,
                                      const float3   r_kl,
                                      const float3   m,
                                      const float3   n,
                                      float3         gm_f[],
                                      float3         sm_fShiftLoc[],
                                      const PbcAiuc& pbcAiuc,
                                      const float4   gm_xq[],
                                      const int      t1,
                                      const int      t2,
                                      const int gmx_unused t3)
{
    float iprm  = norm2(m);
    float iprn  = norm2(n);
    float nrkj2 = norm2(r_kj);
    float toler = nrkj2 * GMX_REAL_EPS;
    if ((iprm > toler) && (iprn > toler))
    {
        float  nrkj_1 = rsqrtf(nrkj2); // replacing std::invsqrt call
        float  nrkj_2 = nrkj_1 * nrkj_1;
        float  nrkj   = nrkj2 * nrkj_1;
        float  a      = -ddphi * nrkj / iprm;
        float3 f_i    = a * m;
        float  b      = ddphi * nrkj / iprn;
        float3 f_l    = b * n;
        float  p      = iprod(r_ij, r_kj);
        p *= nrkj_2;
        float q = iprod(r_kl, r_kj);
        q *= nrkj_2;
        float3 uvec = p * f_i;
        float3 vvec = q * f_l;
        float3 svec = uvec - vvec;
        float3 f_j  = f_i - svec;
        float3 f_k  = f_l + svec;

        atomicAdd(&gm_f[i], f_i);
        atomicAdd(&gm_f[j], -f_j);
        atomicAdd(&gm_f[k], -f_k);
        atomicAdd(&gm_f[l], f_l);

        if (calcVir)
        {
            float3 dx_jl;
            int    t3 = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[l], gm_xq[j], dx_jl);

            atomicAdd(&sm_fShiftLoc[t1], f_i);
            atomicAdd(&sm_fShiftLoc[CENTRAL], -f_j);
            atomicAdd(&sm_fShiftLoc[t2], -f_k);
            atomicAdd(&sm_fShiftLoc[t3], f_l);
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void pdihs_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          const float4    gm_xq[],
                          float3          gm_f[],
                          float3          sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        float3 r_ij;
        float3 r_kj;
        float3 r_kl;
        float3 m;
        float3 n;
        int    t1;
        int    t2;
        int    t3;
        float  phi = dih_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc,
                                           &r_ij, &r_kj, &r_kl, &m, &n, &t1, &t2, &t3);

        float vpd;
        float ddphi;
        dopdihs_gpu(d_forceparams[type].pdihs.cpA, d_forceparams[type].pdihs.phiA,
                    d_forceparams[type].pdihs.mult, phi, &vpd, &ddphi);

        if (calcEner)
        {
            *vtot_loc += vpd;
        }

        do_dih_fup_gpu<calcVir>(ai, aj, ak, al, ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc,
                                pbcAiuc, gm_xq, t1, t2, t3);
    }
}

template<bool calcVir, bool calcEner>
__device__ void pdihs_fep_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          gmx::BondedFepParameters* d_fepparams,
                          const float4    gm_xq[],
                          float3            gm_f[],
                          float3            sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        float3  r_ij;
        float3  r_kj;
        float3  r_kl;
        float3  m;
        float3  n;
        int   t1;
        int   t2;
        int   t3;
        float phi = dih_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc,
                                           &r_ij, &r_kj, &r_kl, &m, &n, &t1, &t2, &t3);

        float vpd;
        float ddphi;
        dopdihs_fep_gpu(d_forceparams[type].pdihs.cpA, d_forceparams[type].pdihs.cpB, d_forceparams[type].pdihs.phiA, d_forceparams[type].pdihs.phiB,
                    d_forceparams[type].pdihs.mult, phi, d_fepparams->lambda_q, &vpd, &ddphi);

        if (calcEner)
        {
            *vtot_loc += vpd;
        }

        do_dih_fup_gpu<calcVir>(ai, aj, ak, al, ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc,
                                pbcAiuc, gm_xq, t1, t2, t3);
    }
}

template<bool calcVir, bool calcEner>
__device__ void rbdihs_gpu(const int       i,
                           float*          vtot_loc,
                           const int       numBonds,
                           const t_iatom   d_forceatoms[],
                           const t_iparams d_forceparams[],
                           const float4    gm_xq[],
                           float3          gm_f[],
                           float3          sm_fShiftLoc[],
                           const PbcAiuc   pbcAiuc)
{
    constexpr float c0 = 0.0f, c1 = 1.0f, c2 = 2.0f, c3 = 3.0f, c4 = 4.0f, c5 = 5.0f;

    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        float3 r_ij;
        float3 r_kj;
        float3 r_kl;
        float3 m;
        float3 n;
        int    t1;
        int    t2;
        int    t3;
        float  phi = dih_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc,
                                           &r_ij, &r_kj, &r_kl, &m, &n, &t1, &t2, &t3);

        /* Change to polymer convention */
        if (phi < c0)
        {
            phi += HIP_PI_F;
        }
        else
        {
            phi -= HIP_PI_F;
        }
        float cos_phi = cosf(phi);
        /* Beware of accuracy loss, cannot use 1-sqrt(cos^2) ! */
        float sin_phi = sinf(phi);

        float parm[NR_RBDIHS];
        for (int j = 0; j < NR_RBDIHS; j++)
        {
            parm[j] = d_forceparams[type].rbdihs.rbcA[j];
        }
        /* Calculate cosine powers */
        /* Calculate the energy */
        /* Calculate the derivative */
        float v      = parm[0];
        float ddphi  = c0;
        float cosfac = c1;

        float rbp = parm[1];
        ddphi += rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[2];
        ddphi += c2 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[3];
        ddphi += c3 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[4];
        ddphi += c4 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[5];
        ddphi += c5 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }

        ddphi = -ddphi * sin_phi;

        do_dih_fup_gpu<calcVir>(ai, aj, ak, al, ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc,
                                pbcAiuc, gm_xq, t1, t2, t3);
        if (calcEner)
        {
            *vtot_loc += v;
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void rbdihs_fep_gpu(const int       i,
                           float*          vtot_loc,
                           const int       numBonds,
                           const t_iatom   d_forceatoms[],
                           const t_iparams d_forceparams[],
                           gmx::BondedFepParameters* d_fepparams,
                           const float4    gm_xq[],
                           float3            gm_f[],
                           float3            sm_fShiftLoc[],
                           const PbcAiuc   pbcAiuc)
{
    constexpr float c0 = 0.0f, c1 = 1.0f, c2 = 2.0f, c3 = 3.0f, c4 = 4.0f, c5 = 5.0f;

    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        float3  r_ij;
        float3  r_kj;
        float3  r_kl;
        float3  m;
        float3  n;
        int   t1;
        int   t2;
        int   t3;
        float phi = dih_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc,
                                           &r_ij, &r_kj, &r_kl, &m, &n, &t1, &t2, &t3);
        float lambda = d_fepparams->lambda_q;

        /* Change to polymer convention */
        if (phi < c0)
        {
            phi += HIP_PI_F;
        }
        else
        {
            phi -= HIP_PI_F;
        }
        float cos_phi = cosf(phi);
        /* Beware of accuracy loss, cannot use 1-sqrt(cos^2) ! */
        float sin_phi = sinf(phi);

        float parm[NR_RBDIHS];
        for (int j = 0; j < NR_RBDIHS; j++)
        {
            parm[j] = (1 - lambda) * d_forceparams[type].rbdihs.rbcA[j] + lambda * d_forceparams[type].rbdihs.rbcB[j];
        }
        /* Calculate cosine powers */
        /* Calculate the energy */
        /* Calculate the derivative */
        float v      = parm[0];
        float ddphi  = c0;
        float cosfac = c1;

        float rbp = parm[1];
        ddphi += rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[2];
        ddphi += c2 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[3];
        ddphi += c3 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[4];
        ddphi += c4 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }
        rbp = parm[5];
        ddphi += c5 * rbp * cosfac;
        cosfac *= cos_phi;
        if (calcEner)
        {
            v += cosfac * rbp;
        }

        ddphi = -ddphi * sin_phi;

        do_dih_fup_gpu<calcVir>(ai, aj, ak, al, ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc,
                                pbcAiuc, gm_xq, t1, t2, t3);
        if (calcEner)
        {
            *vtot_loc += v;
        }
    }
}

__device__ __forceinline__ static void make_dp_periodic_gpu(float* dp)
{
    /* dp cannot be outside (-pi,pi) */
    if (*dp >= HIP_PI_F)
    {
        *dp -= 2.0f * HIP_PI_F;
    }
    else if (*dp < -HIP_PI_F)
    {
        *dp += 2.0f * HIP_PI_F;
    }
}

template<bool calcVir, bool calcEner>
__device__ void idihs_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          const float4    gm_xq[],
                          float3          gm_f[],
                          float3          sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        float3 r_ij;
        float3 r_kj;
        float3 r_kl;
        float3 m;
        float3 n;
        int    t1;
        int    t2;
        int    t3;
        float  phi = dih_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc,
                                           &r_ij, &r_kj, &r_kl, &m, &n, &t1, &t2, &t3);

        /* phi can jump if phi0 is close to Pi/-Pi, which will cause huge
         * force changes if we just apply a normal harmonic.
         * Instead, we first calculate phi-phi0 and take it modulo (-Pi,Pi).
         * This means we will never have the periodicity problem, unless
         * the dihedral is Pi away from phiO, which is very unlikely due to
         * the potential.
         */
        float kA = d_forceparams[type].harmonic.krA;
        float pA = d_forceparams[type].harmonic.rA;

        float phi0 = pA * CUDA_DEG2RAD_F;

        float dp = phi - phi0;

        make_dp_periodic_gpu(&dp);

        float ddphi = -kA * dp;

        do_dih_fup_gpu<calcVir>(ai, aj, ak, al, -ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc,
                                pbcAiuc, gm_xq, t1, t2, t3);

        if (calcEner)
        {
            *vtot_loc += -0.5f * ddphi * dp;
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void idihs_fep_gpu(const int       i,
                          float*          vtot_loc,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams d_forceparams[],
                          gmx::BondedFepParameters* d_fepparams,
                          const float4    gm_xq[],
                          float3            gm_f[],
                          float3            sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc)
{
    if (i < numBonds)
    {
        int type = d_forceatoms[5 * i];
        int ai   = d_forceatoms[5 * i + 1];
        int aj   = d_forceatoms[5 * i + 2];
        int ak   = d_forceatoms[5 * i + 3];
        int al   = d_forceatoms[5 * i + 4];

        float3  r_ij;
        float3  r_kj;
        float3  r_kl;
        float3  m;
        float3  n;
        int   t1;
        int   t2;
        int   t3;
        float phi = dih_angle_gpu<calcVir>(gm_xq[ai], gm_xq[aj], gm_xq[ak], gm_xq[al], pbcAiuc,
                                           &r_ij, &r_kj, &r_kl, &m, &n, &t1, &t2, &t3);

        /* phi can jump if phi0 is close to Pi/-Pi, which will cause huge
         * force changes if we just apply a normal harmonic.
         * Instead, we first calculate phi-phi0 and take it modulo (-Pi,Pi).
         * This means we will never have the periodicity problem, unless
         * the dihedral is Pi away from phiO, which is very unlikely due to
         * the potential.
         */
        float kA = d_forceparams[type].harmonic.krA;
        float pA = d_forceparams[type].harmonic.rA;
        float kB = d_forceparams[type].harmonic.krB;
        float pB = d_forceparams[type].harmonic.rB;

        float phi0A = pA * CUDA_DEG2RAD_F;
        float phi0B = pB * CUDA_DEG2RAD_F;

        float dpA = phi - phi0A;
        float dpB = phi - phi0B;

        make_dp_periodic_gpu(&dpA);
        make_dp_periodic_gpu(&dpB);

        float lambda = d_fepparams->lambda_q;

        float ddphi = -((1 - lambda) * kA * dpA + lambda * kB * dpB);

        do_dih_fup_gpu<calcVir>(ai, aj, ak, al, -ddphi, r_ij, r_kj, r_kl, m, n, gm_f, sm_fShiftLoc,
                                pbcAiuc, gm_xq, t1, t2, t3);

        if (calcEner)
        {
            *vtot_loc += -0.5f * ddphi * dpA;
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void pairs_gpu(const int       i,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams iparams[],
                          const float4    gm_xq[],
                          float3          gm_f[],
                          float3          sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc,
                          const float     scale_factor,
                          float*          vtotVdw_loc,
                          float*          vtotElec_loc)
{
    if (i < numBonds)
    {
        // TODO this should be made into a separate type, the GPU and CPU sizes should be compared
        int3 pairData = *(int3*)(d_forceatoms + 3 * i);
        int  type     = pairData.x;
        int  ai       = pairData.y;
        int  aj       = pairData.z;

        float qq  = gm_xq[ai].w * gm_xq[aj].w;
        float c6  = iparams[type].lj14.c6A;
        float c12 = iparams[type].lj14.c12A;

        /* Do we need to apply full periodic boundary conditions? */
        float3 dr;
        int    fshift_index = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[aj], dr);

        float r2    = norm2(dr);
        float rinv  = rsqrtf(r2);
        float rinv2 = rinv * rinv;
        float rinv6 = rinv2 * rinv2 * rinv2;

        /* Calculate the Coulomb force * r */
        float velec = scale_factor * qq * rinv;

        /* Calculate the LJ force * r and add it to the Coulomb part */
        float fr = (12.0f * c12 * rinv6 - 6.0f * c6) * rinv6 + velec;

        float  finvr = fr * rinv2;
        float3 f     = finvr * dr;

        /* Add the forces */
        atomicAdd(&gm_f[ai], f);
        atomicAdd(&gm_f[aj], -f);
        if (calcVir && fshift_index != CENTRAL)
        {
            atomicAdd(&sm_fShiftLoc[fshift_index], f);
            atomicAdd(&sm_fShiftLoc[CENTRAL], -f);
        }

        if (calcEner)
        {
            *vtotVdw_loc += (c12 * rinv6 - c6) * rinv6;
            *vtotElec_loc += velec;
        }
    }
}

template<bool calcVir, bool calcEner>
__device__ void pairs_fep_gpu(const int       i,
                          const int       numBonds,
                          const t_iatom   d_forceatoms[],
                          const t_iparams iparams[],
                          gmx::BondedFepParameters* d_fepparams,
                          const float4    gm_xq[],
                          const float     gm_qA[],
                          const float     gm_qB[],
                          float3            gm_f[],
                          float3            sm_fShiftLoc[],
                          const PbcAiuc   pbcAiuc,
                          const float     scale_factor,
                          float*          vtotVdw_loc,
                          float*          vtotElec_loc)
{
    const bool bFEP = d_fepparams->bFEP;
    bool       bFEPpair = 0;
    const float alpha_coul = d_fepparams->alpha_coul;
    const float alpha_vdw  = d_fepparams->alpha_vdw;
    float alpha_coul_eff   = alpha_coul;
    float alpha_vdw_eff    = alpha_vdw;
    const bool useSoftCore = (alpha_vdw != 0.0);
    const bool useScBetaNO = (alpha_coul == 0.0);
    const float sigma6_def = d_fepparams->sc_sigma6;
    const float sigma6_min = d_fepparams->sc_sigma6_min;
    const float lambda_q   = d_fepparams->lambda_q;
    const float _lambda_q  = 1 - lambda_q;
    const float lambda_v   = d_fepparams->lambda_v;
    const float _lambda_v  = 1 - lambda_v;

    const float lfac_coul[2] = {lambda_q, _lambda_q};
    const float lfac_vdw[2]  = {lambda_v, _lambda_v};
    const float LFC[2]       = {_lambda_q, lambda_q};
    const float LFV[2]       = {_lambda_v, lambda_v};

    float FscalC[2], FscalV[2], Vcoul[2], Vvdw[2];
    float rinvC, rinvV, r2C, r2V, rpinvC, rpinvV;

    if (i < numBonds)
    {
        int3 pairData = *(int3*)(d_forceatoms + 3 * i);
        int  type     = pairData.x;
        int  ai       = pairData.y;
        int  aj       = pairData.z;

        float qq[2]  = {gm_qA[ai] * gm_qA[aj], gm_qB[ai] * gm_qB[aj]};
        float c6AB[2]  = {iparams[type].lj14.c6A, iparams[type].lj14.c6B};
        float c12AB[2] = {iparams[type].lj14.c12A, iparams[type].lj14.c12B};
        float sigma6[2];
        float velec = 0;
        float vlj   = 0;
        float finvr = 0;

        if (qq[0] == qq[1] && c6AB[0] == c6AB[1] && c12AB[0] == c12AB[1]) bFEPpair = 0;
        else bFEPpair = 1;

        /* Do we need to apply full periodic boundary conditions? */
        float3 dr;
        int  fshift_index = pbcDxAiuc<calcVir>(pbcAiuc, gm_xq[ai], gm_xq[aj], dr);

        float r2    = norm2(dr);
        float rpm2  = r2 * r2;
        float rp    = rpm2 * r2;
        float rinv  = rsqrtf(r2);
        float rinv2 = rinv * rinv;
        float rinv6 = rinv2 * rinv2 * rinv2;

        if (bFEPpair)
        {
            for (int k = 0; k < 2; k++)
            {
                FscalC[k] = 0;
                FscalV[k] = 0;
                if (calcEner)
                {
                    Vcoul[k] = 0;
                    Vvdw[k]  = 0;
                }
                if ((qq[k] != 0) || (c6AB[k] != 0) || (c12AB[k] != 0))
                {
                    if ((c12AB[0] == 0 || c12AB[1] == 0) && (useSoftCore))
                    {
                        if (c6AB[k] == 0)
                            sigma6[k] = 0;
                        else
                            sigma6[k] = c12AB[k] / c6AB[k];
                        alpha_vdw_eff  = alpha_vdw;
                        alpha_coul_eff = (useScBetaNO ? alpha_vdw_eff : alpha_coul);
                        if (sigma6[k] == 0)
                            sigma6[k] = sigma6_def;
                        if (sigma6[k] < sigma6_min)
                            sigma6[k] = sigma6_min;

                        if (useScBetaNO)
                        {
                            rpinvC = 1.0f / (alpha_coul_eff * lfac_coul[k] * sigma6[k] + rp);
                        }
                        // else if (softCoreTreatment == SoftCoreTreatment::RPower6_2)
                        // {
                        //     rpinvC = one / (alpha_coul_eff * lfac_coul[i] + rpc);
                        //     sqRoot<softCoreTreatment>(rpinvC, &rinvC, &rC);
                        // }
                        else
                        {
                            rpinvC = 1.0f / (alpha_coul_eff * lfac_coul[k] * sigma6[k] + rp);
                        }
                        r2C   = rcbrt(rpinvC);
                        rinvC = rsqrt(r2C);

                        if ((alpha_coul_eff != alpha_vdw_eff) || (!useScBetaNO))
                        {
                            rpinvV = 1.0f / (alpha_vdw_eff * lfac_vdw[k] * sigma6[k] + rp);
                            r2V    = rcbrt(rpinvV);
                            rinvV  = rsqrt(r2V);
                        }
                        else
                        {
                            /* We can avoid one expensive pow and one / operation */
                            rpinvV = rpinvC;
                            rinvV  = rinvC;
                        }
                    }
                    else
                    {
                        rpinvC = rinv6;
                        rinvC  = rinv;
                        rpinvV = rinv6;
                        rinvV  = rinv;
                    }
                    // #        endif /* EXCLUSION_FORCES */
                    // #    if !defined LJ_COMB_LB || defined CALC_ENERGIES

                    if (c6AB[k] != 0 || c12AB[k] != 0)
                    {
                        float Vvdw6  = c6AB[k] * rpinvV;
                        float Vvdw12 = c12AB[k] * rpinvV * rpinvV;
                        FscalV[k]    = 12.0f * Vvdw12 - 6.0f * Vvdw6;
                        if (calcEner)
                        {
                            Vvdw[k] = Vvdw12 - Vvdw6;
                        }
                    }

                    if (qq[k] != 0)
                    {
                        FscalC[k] = scale_factor * qq[k] * rinvC;
                        Vcoul[k]  = FscalC[k];
                    }
                    // if ((c6AB[0] != 0 || c6AB[1] != 0 || c12AB[0]!=0 || c12AB[1]!=0 || qq[0]!=0 || qq[1]!=0) && (k == 1))
                    // printf("interaction [%d-%d], r2=[%e], mask=%f, FscalC=[%e, %e], FscalV=[%e, %e]\n", ai, aj, r2, int_bit, FscalC[0] * rpm2, FscalC[1] * rpm2, FscalV[0] * rpm2, FscalV[1] * rpm2);
                    FscalC[k] *= rpinvC;
                    FscalV[k] *= rpinvV;
                }
            }
            for (int k = 0; k < 2; k++)
            {
                if (calcEner)
                {
                    velec += LFC[k] * Vcoul[k];
                    vlj += LFV[k] * Vvdw[k];
                }
                finvr += LFC[k] * FscalC[k] * rpm2;
                finvr += LFV[k] * FscalV[k] * rpm2;
            }
        }
        else
        {
            /* Calculate the Coulomb force * r */
            velec = scale_factor * qq[0] * rinv;
            vlj   = (c12AB[0] * rinv6 - c6AB[0]) * rinv6;

            /* Calculate the LJ force * r and add it to the Coulomb part */
            float fr = (12.0f * c12AB[0] * rinv6 - 6.0f * c6AB[0]) * rinv6 + velec;
            finvr    = fr * rinv2;
        }

        float3 f = finvr * dr;

        /* Add the forces */
        atomicAdd(&gm_f[ai], f);
        atomicAdd(&gm_f[aj], -f);
        if (calcVir && fshift_index != CENTRAL)
        {
            atomicAdd(&sm_fShiftLoc[fshift_index], f);
            atomicAdd(&sm_fShiftLoc[CENTRAL], -f);
        }

        if (calcEner)
        {
            *vtotVdw_loc += vlj;
            *vtotElec_loc += velec;
        }
    }
}

namespace gmx
{

template<bool calcVir, bool calcEner>
__global__ void exec_kernel_gpu(BondedCudaKernelParameters kernelParams)
{
    assert(blockDim.y == 1 && blockDim.z == 1);
    const int tid          = blockIdx.x * blockDim.x + threadIdx.x;
    float     vtot_loc     = 0;
    float     vtotVdw_loc  = 0;
    float     vtotElec_loc = 0;

    extern __shared__ char sm_dynamicShmem[];
    char*                  sm_nextSlotPtr = sm_dynamicShmem;
    float3*                sm_fShiftLoc   = (float3*)sm_nextSlotPtr;
    sm_nextSlotPtr += SHIFTS * sizeof(float3);

    if (calcVir)
    {
        if (threadIdx.x < SHIFTS)
        {
            sm_fShiftLoc[threadIdx.x] = make_float3(0.0f, 0.0f, 0.0f);
        }
        __syncthreads();
    }

    int  fType;
    bool threadComputedPotential = false;
#pragma unroll
    for (int j = 0; j < numFTypesOnGpu; j++)
    {
        if (tid >= kernelParams.fTypeRangeStart[j] && tid <= kernelParams.fTypeRangeEnd[j])
        {
            const int      numBonds = kernelParams.numFTypeBonds[j];
            int            fTypeTid = tid - kernelParams.fTypeRangeStart[j];
            const t_iatom* iatoms   = kernelParams.d_iatoms[j];
            fType                   = kernelParams.fTypesOnGpu[j];
            if (calcEner)
            {
                threadComputedPotential = true;
            }

            switch (fType)
            {
                case F_BONDS:
                    // bonds_gpu<calcVir, calcEner>(fTypeTid, &vtot_loc, numBonds, iatoms,
                    //                              kernelParams.d_forceParams, kernelParams.d_xq,
                    //                              kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    bonds_fep_gpu<calcVir, calcEner>(fTypeTid, &vtot_loc, numBonds, iatoms,
                                                 kernelParams.d_forceParams, kernelParams.d_fepParams, kernelParams.d_xq,
                                                 kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_ANGLES:
                    // angles_gpu<calcVir, calcEner>(
                    //         fTypeTid, &vtot_loc, numBonds, iatoms, kernelParams.d_forceParams,
                    //         kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    angles_fep_gpu<calcVir, calcEner>(
                            fTypeTid, &vtot_loc, numBonds, iatoms, kernelParams.d_forceParams, kernelParams.d_fepParams,
                            kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_UREY_BRADLEY:
                    // urey_bradley_gpu<calcVir, calcEner>(
                    //         fTypeTid, &vtot_loc, numBonds, iatoms, kernelParams.d_forceParams,
                    //         kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    urey_bradley_fep_gpu<calcVir, calcEner>(
                            fTypeTid, &vtot_loc, numBonds, iatoms, kernelParams.d_forceParams, kernelParams.d_fepParams,
                            kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_PDIHS:
                case F_PIDIHS:
                    // pdihs_gpu<calcVir, calcEner>(fTypeTid, &vtot_loc, numBonds, iatoms,
                    //                              kernelParams.d_forceParams, kernelParams.d_xq,
                    //                              kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    pdihs_fep_gpu<calcVir, calcEner>(fTypeTid, &vtot_loc, numBonds, iatoms,
                                                 kernelParams.d_forceParams, kernelParams.d_fepParams, kernelParams.d_xq,
                                                 kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_RBDIHS:
                    // rbdihs_gpu<calcVir, calcEner>(
                    //         fTypeTid, &vtot_loc, numBonds, iatoms, kernelParams.d_forceParams,
                    //         kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    rbdihs_fep_gpu<calcVir, calcEner>(
                            fTypeTid, &vtot_loc, numBonds, iatoms, kernelParams.d_forceParams, kernelParams.d_fepParams,
                            kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_IDIHS:
                    // idihs_gpu<calcVir, calcEner>(fTypeTid, &vtot_loc, numBonds, iatoms,
                    //                              kernelParams.d_forceParams, kernelParams.d_xq,
                    //                              kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    idihs_fep_gpu<calcVir, calcEner>(fTypeTid, &vtot_loc, numBonds, iatoms,
                                                 kernelParams.d_forceParams, kernelParams.d_fepParams, kernelParams.d_xq,
                                                 kernelParams.d_f, sm_fShiftLoc, kernelParams.pbcAiuc);
                    break;
                case F_LJ14:
                    // pairs_gpu<calcVir, calcEner>(fTypeTid, numBonds, iatoms, kernelParams.d_forceParams,
                    //                              kernelParams.d_xq, kernelParams.d_f, sm_fShiftLoc,
                    //                              kernelParams.pbcAiuc, kernelParams.electrostaticsScaleFactor,
                    //                              &vtotVdw_loc, &vtotElec_loc);
                    pairs_fep_gpu<calcVir, calcEner>(fTypeTid, numBonds, iatoms, kernelParams.d_forceParams, kernelParams.d_fepParams,
                                                 kernelParams.d_xq, kernelParams.d_qA, kernelParams.d_qB, kernelParams.d_f, sm_fShiftLoc,
                                                 kernelParams.pbcAiuc, kernelParams.electrostaticsScaleFactor,
                                                 &vtotVdw_loc, &vtotElec_loc);
                    break;
            }
            break;
        }
    }

    if (threadComputedPotential)
    {
        float* vtotVdw  = kernelParams.d_vTot + F_LJ14;
        float* vtotElec = kernelParams.d_vTot + F_COUL14;

        // Stage atomic accumulation through shared memory:
        // each warp will accumulate its own partial sum
        // and then a single thread per warp will accumulate this to the global sum

        int numWarps = blockDim.x / warpSize;
        int warpId   = threadIdx.x / warpSize;

        // Shared memory variables to hold block-local partial sum
        float* sm_vTot = (float*)sm_nextSlotPtr;
        sm_nextSlotPtr += numWarps * sizeof(float);
        float* sm_vTotVdw = (float*)sm_nextSlotPtr;
        sm_nextSlotPtr += numWarps * sizeof(float);
        float* sm_vTotElec = (float*)sm_nextSlotPtr;

        if (threadIdx.x % warpSize == 0)
        {
            // One thread per warp initializes to zero
            sm_vTot[warpId]     = 0.;
            sm_vTotVdw[warpId]  = 0.;
            sm_vTotElec[warpId] = 0.;
        }
        __syncwarp(); // All threads in warp must wait for initialization

        // Perform warp-local accumulation in shared memory
        atomicAdd(sm_vTot + warpId, vtot_loc);
        atomicAdd(sm_vTotVdw + warpId, vtotVdw_loc);
        atomicAdd(sm_vTotElec + warpId, vtotElec_loc);

        __syncwarp(); // Ensure all threads in warp have completed
        if (threadIdx.x % warpSize == 0)
        { // One thread per warp accumulates partial sum into global sum
            atomicAdd(kernelParams.d_vTot + fType, sm_vTot[warpId]);
            atomicAdd(vtotVdw, sm_vTotVdw[warpId]);
            atomicAdd(vtotElec, sm_vTotElec[warpId]);
        }
    }
    /* Accumulate shift vectors from shared memory to global memory on the first SHIFTS threads of the block. */
    if (calcVir)
    {
        __syncthreads();
        if (threadIdx.x < SHIFTS)
        {
            atomicAdd(kernelParams.d_fShift[threadIdx.x], sm_fShiftLoc[threadIdx.x]);
        }
    }
}


/*-------------------------------- End CUDA kernels-----------------------------*/


template<bool calcVir, bool calcEner>
void GpuBonded::Impl::launchKernel()
{
    GMX_ASSERT(haveInteractions_,
               "Cannot launch bonded GPU kernels unless bonded GPU work was scheduled");

    wallcycle_start_nocount(wcycle_, ewcLAUNCH_GPU);
    wallcycle_sub_start(wcycle_, ewcsLAUNCH_GPU_BONDED);

    int fTypeRangeEnd = kernelParams_.fTypeRangeEnd[numFTypesOnGpu - 1];

    if (fTypeRangeEnd < 0)
    {
        return;
    }

    auto kernelPtr = exec_kernel_gpu<calcVir, calcEner>;

    const auto kernelArgs = prepareGpuKernelArguments(kernelPtr, kernelLaunchConfig_, &kernelParams_);

    launchGpuKernel(kernelPtr, kernelLaunchConfig_, deviceStream_, nullptr,
                    "exec_kernel_gpu<calcVir, calcEner>", kernelArgs);

    wallcycle_sub_stop(wcycle_, ewcsLAUNCH_GPU_BONDED);
    wallcycle_stop(wcycle_, ewcLAUNCH_GPU);
}

void GpuBonded::launchKernel(const gmx::StepWorkload& stepWork)
{
    if (stepWork.computeEnergy)
    {
        // When we need the energy, we also need the virial
        impl_->launchKernel<true, true>();
    }
    else if (stepWork.computeVirial)
    {
        impl_->launchKernel<true, false>();
    }
    else
    {
        impl_->launchKernel<false, false>();
    }
}

} // namespace gmx
